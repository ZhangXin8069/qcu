#include "hip/hip_runtime.h"
#pragma optimize(5)
#include "../../include/qcu.h"

__global__ void give_random_value(void *device_random_value,
                                  unsigned long seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lat_4dim = gridDim.x * blockDim.x;
  if (idx < lat_4dim) {
    LatticeComplex *random_value =
        static_cast<LatticeComplex *>(device_random_value);
    hiprandState state_real, state_imag;
    hiprand_init(seed, idx, 0, &state_real);
    hiprand_init(seed, idx, 1, &state_imag);
    for (int i = 0; i < LAT_SC; ++i) {
      random_value[idx * LAT_SC + i].real = hiprand_uniform(&state_real);
      random_value[idx * LAT_SC + i].imag = hiprand_uniform(&state_imag);
    }
  }
}

__global__ void give_custom_value(void *device_custom_value, double real,
                                  double imag) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lat_4dim = gridDim.x * blockDim.x;
  if (idx < lat_4dim) {
    LatticeComplex *custom_value =
        static_cast<LatticeComplex *>(device_custom_value);
    for (int i = 0; i < LAT_SC; ++i) {
      custom_value[idx * LAT_SC + i].real = real;
      custom_value[idx * LAT_SC + i].imag = imag;
    }
  }
}