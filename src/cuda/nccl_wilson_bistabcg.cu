#include "../../include/qcu.h"
#ifdef NCCL_WILSON_BISTABCG
// #define DEBUG_NCCL_WILSON_CG
void ncclBistabCgQcu(void *gauge, QcuParam *param, QcuParam *grid) {
  int node_rank, node_size, localRank = 0;
  // initializing MPI
  // MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &node_rank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &node_size));
  // calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[node_size];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[node_rank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                         sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p = 0; p < node_size; p++) {
    if (p == node_rank)
      break;
    if (hostHashs[p] == hostHashs[node_rank])
      localRank++;
  }
  ncclUniqueId nccl_id;
  ncclComm_t nccl_comm;
  hipStream_t stream;
  // get NCCL unique nccl_id at rank 0 and broadcast it to all others
  if (node_rank == 0)
    ncclGetUniqueId(&nccl_id);
  MPICHECK(MPI_Bcast((void *)&nccl_id, sizeof(nccl_id), MPI_BYTE, 0,
                     MPI_COMM_WORLD));
  // picking a GPU based on localRank
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipStreamCreate(&stream));
  // initializing NCCL
  NCCLCHECK(ncclCommInitRank(&nccl_comm, node_size, nccl_id, node_rank));
  // define for nccl_wilson_dslash
  int lat_1dim[_DIM_];
  int lat_3dim[_DIM_];
  int lat_4dim;
  give_dims(param, lat_1dim, lat_3dim, lat_4dim);
  int lat_3dim6[_DIM_];
  int lat_3dim12[_DIM_];
  for (int i = 0; i < _DIM_; i++) {
    lat_3dim6[i] = lat_3dim[i] * 6;
    lat_3dim12[i] = lat_3dim6[i] * 2;
  }
  hipError_t err;
  dim3 gridDim(lat_4dim / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  int move[_BF_];
  int grid_1dim[_DIM_];
  int grid_index_1dim[_DIM_];
  give_grid(grid, node_rank, grid_1dim, grid_index_1dim);
  void *host_send_vec[_WARDS_];
  void *host_recv_vec[_WARDS_];
  void *device_send_vec[_WARDS_];
  void *device_recv_vec[_WARDS_];
  malloc_vec(lat_3dim6, device_send_vec, device_recv_vec, host_send_vec,
             host_recv_vec);
  // define end
  // define for nccl_wilson_cg
  int lat_4dim12 = lat_4dim * 12;
  LatticeComplex r_norm2(0.0, 0.0);
  const int MAX_ITER(1e3); // 300++?
  const double TOL(1e-6);
  LatticeComplex rho_prev(1.0, 0.0);
  LatticeComplex rho(0.0, 0.0);
  LatticeComplex alpha(1.0, 0.0);
  LatticeComplex omega(1.0, 0.0);
  LatticeComplex beta(0.0, 0.0);
  double kappa = 0.125;
  LatticeComplex tmp(0.0, 0.0);
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex local_result(0.0, 0.0);
  void *ans_e, *ans_o, *x_e, *x_o, *b_e, *b_o, *b__o, *r, *r_tilde, *p, *v, *s,
      *t, *device_latt_tmp0, *device_latt_tmp1, *device_dot_tmp;
  hipMalloc(&ans_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&ans_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&x_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&x_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b__o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&r, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&r_tilde, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&p, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&v, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&s, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&t, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&device_latt_tmp0, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&device_latt_tmp1, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&device_dot_tmp, lat_4dim * sizeof(LatticeComplex));
  LatticeComplex *host_latt_tmp0 =
      (LatticeComplex *)malloc(lat_4dim12 * sizeof(LatticeComplex));
  LatticeComplex *host_latt_tmp1 =
      (LatticeComplex *)malloc(lat_4dim12 * sizeof(LatticeComplex));
  LatticeComplex *host_dot_tmp =
      (LatticeComplex *)malloc(lat_4dim * sizeof(LatticeComplex));
  // give ans first
  give_random_value<<<gridDim, blockDim>>>(ans_e, node_rank + 12138);
  give_random_value<<<gridDim, blockDim>>>(ans_o, node_rank + 83121);
  // give x_o, b_e, b_o ,b__o, r, r_tilde, p, v, s, t
  give_random_value<<<gridDim, blockDim>>>(x_o, node_rank + 66666);
  give_custom_value<<<gridDim, blockDim>>>(b_e, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(b_o, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(b__o, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(r, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(r_tilde, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(p, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(v, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(s, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(t, 0.0, 0.0);
  // give b'_o(b__0)
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp0, 0.0, 0.0);
  checkCudaErrors(hipDeviceSynchronize());
  nccl_dslash_eo(device_latt_tmp0, ans_o, node_rank, gridDim, blockDim, gauge,
                 lat_1dim, lat_3dim12, grid_1dim, grid_index_1dim, move,
                 device_send_vec, device_recv_vec, nccl_comm, stream);
  wilson_bistabcg_give_b_e<<<gridDim, blockDim>>>(b_e, ans_e, device_latt_tmp0,
                                                  kappa);
  checkCudaErrors(hipDeviceSynchronize());
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp1, 0.0, 0.0);
  checkCudaErrors(hipDeviceSynchronize());
  nccl_dslash_oe(device_latt_tmp1, ans_e, node_rank, gridDim, blockDim, gauge,
                 lat_1dim, lat_3dim12, grid_1dim, grid_index_1dim, move,
                 device_send_vec, device_recv_vec, nccl_comm, stream);
  wilson_bistabcg_give_b_o<<<gridDim, blockDim>>>(b_o, ans_o, device_latt_tmp1,
                                                  kappa);
  checkCudaErrors(hipDeviceSynchronize());
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp0, 0.0, 0.0);
  checkCudaErrors(hipDeviceSynchronize());
  nccl_dslash_oe(device_latt_tmp0, b_e, node_rank, gridDim, blockDim, gauge,
                 lat_1dim, lat_3dim12, grid_1dim, grid_index_1dim, move,
                 device_send_vec, device_recv_vec, nccl_comm, stream);
  wilson_bistabcg_give_b__0<<<gridDim, blockDim>>>(b__o, b_o, device_latt_tmp0,
                                                   kappa);
  checkCudaErrors(hipDeviceSynchronize());
  // bistabcg
  nccl_dslash(r, x_o, kappa, device_latt_tmp0, device_latt_tmp1, node_rank,
              gridDim, blockDim, gauge, lat_1dim, lat_3dim12, lat_4dim12,
              grid_1dim, grid_index_1dim, move, device_send_vec,
              device_recv_vec, nccl_comm, stream);
  wilson_bistabcg_give_rr<<<gridDim, blockDim>>>(r, b__o, r_tilde);
  checkCudaErrors(hipDeviceSynchronize());
  // define end
  auto start = std::chrono::high_resolution_clock::now();
  for (int loop = 0; loop < MAX_ITER; loop++) {
    nccl_dot(device_dot_tmp, host_dot_tmp, r_tilde, r, rho, gridDim, blockDim);
#ifdef DEBUG_NCCL_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##rho:" << rho.real << std::endl;
#endif
    beta = (rho / rho_prev) * (alpha / omega);
#ifdef DEBUG_NCCL_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##beta:" << beta.real << std::endl;
#endif
    wilson_bistabcg_give_p<<<gridDim, blockDim>>>(p, r, v, omega, beta);
    checkCudaErrors(hipDeviceSynchronize());
    // v = A * p;
    nccl_dslash(v, p, kappa, device_latt_tmp0, device_latt_tmp1, node_rank,
                gridDim, blockDim, gauge, lat_1dim, lat_3dim12, lat_4dim12,
                grid_1dim, grid_index_1dim, move, device_send_vec,
                device_recv_vec, nccl_comm, stream);
    nccl_dot(device_dot_tmp, host_dot_tmp, r_tilde, v, tmp, gridDim, blockDim);
    alpha = rho / tmp;
#ifdef DEBUG_NCCL_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##alpha:" << alpha.real << std::endl;
#endif
    wilson_bistabcg_give_s<<<gridDim, blockDim>>>(s, r, v, alpha);
    checkCudaErrors(hipDeviceSynchronize());
    // t = A * s;
    nccl_dslash(t, s, kappa, device_latt_tmp0, device_latt_tmp1, node_rank,
                gridDim, blockDim, gauge, lat_1dim, lat_3dim12, lat_4dim12,
                grid_1dim, grid_index_1dim, move, device_send_vec,
                device_recv_vec, nccl_comm, stream);
    nccl_dot(device_dot_tmp, host_dot_tmp, t, s, tmp0, gridDim, blockDim);
    nccl_dot(device_dot_tmp, host_dot_tmp, t, t, tmp1, gridDim, blockDim);
    omega = tmp0 / tmp1;
#ifdef DEBUG_NCCL_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##omega:" << omega.real << std::endl;
#endif
    wilson_bistabcg_give_x_o<<<gridDim, blockDim>>>(x_o, p, s, alpha, omega);
    wilson_bistabcg_give_r<<<gridDim, blockDim>>>(r, s, t, omega);
    checkCudaErrors(hipDeviceSynchronize());
    nccl_dot(device_dot_tmp, host_dot_tmp, r, r, r_norm2, gridDim, blockDim);
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##Residual:" << r_norm2.real << std::endl;
    // break;
    if (r_norm2.real < TOL || loop == MAX_ITER - 1) {
      break;
    }
    rho_prev = rho;
  }
  checkCudaErrors(hipDeviceSynchronize());
  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  err = hipGetLastError();
  checkCudaErrors(err);
  printf("nccl wilson bistabcg total time: (without malloc free "
         "memcpy) :%.9lf "
         "sec\n",
         double(duration) / 1e9);
  nccl_diff(device_dot_tmp, host_dot_tmp, x_o, ans_o, tmp, device_latt_tmp0,
            tmp0, tmp1, gridDim, blockDim);
  printf("## difference: %.16f\n", tmp.real);
  // free
  free_vec(device_send_vec, device_recv_vec, host_send_vec, host_recv_vec);
  hipFree(ans_e);
  hipFree(ans_o);
  hipFree(x_o);
  hipFree(b__o);
  hipFree(r);
  hipFree(r_tilde);
  hipFree(p);
  hipFree(v);
  hipFree(s);
  hipFree(t);
  hipFree(device_latt_tmp0);
  hipFree(device_latt_tmp1);
  free(host_latt_tmp0);
  free(host_latt_tmp1);
}
#endif