#include "../../include/qcu.h"
#include "../../include/qcu_cuda.h"
#include <chrono>
#include <cstdio>

void dslashQcu(void *fermion_out, void *fermion_in, void *gauge,
               QcuParam *param, int parity) {
  const int lat_x = param->lattice_size[0] >> 1;
  const int lat_y = param->lattice_size[1];
  const int lat_z = param->lattice_size[2];
  const int lat_t = param->lattice_size[3];
  void *clover;
  checkCudaErrors(hipMalloc(&clover, (lat_t * lat_z * lat_y * lat_x * 144) *
                                          sizeof(LatticeComplex)));
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // wilson dslash
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    wilson_dslash<<<gridDim, blockDim>>>(gauge, fermion_in, fermion_out, lat_x,
                                         lat_y, lat_z, lat_t, parity);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf(
        "wilson dslash total time: (without malloc free memcpy) : %.9lf sec\n",
        double(duration) / 1e9);
  }
  {
    // make clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    make_clover<<<gridDim, blockDim>>>(gauge, clover, lat_x, lat_y, lat_z,
                                       lat_t, parity);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("make clover total time: (without malloc free memcpy) :%.9lf sec\n ",
           double(duration) / 1e9);
  }
  {
    // inverse clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    inverse_clover<<<gridDim, blockDim>>>(clover, lat_x, lat_y, lat_z);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf(
        "inverse clover total time: (without malloc free memcpy) :%.9lf sec\n ",
        double(duration) / 1e9);
  }
  {
    // give clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    give_clover<<<gridDim, blockDim>>>(clover, fermion_out, lat_x, lat_y,
                                       lat_z);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("give clover total time: (without malloc free memcpy) :%.9lf sec\n ",
           double(duration) / 1e9);
  }
  {
    // free
    checkCudaErrors(hipFree(clover));
  }
}

void testDslashQcu(void *fermion_out, void *fermion_in, void *gauge,
                   QcuParam *param, int parity) {
  const int lat_x = param->lattice_size[0] >> 1;
  const int lat_y = param->lattice_size[1];
  const int lat_z = param->lattice_size[2];
  const int lat_t = param->lattice_size[3];
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // wilson dslash
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    test_wilson_dslash<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("test wilson dslash total time: (without malloc free memcpy) : "
           "%.9lf sec\n",
           double(duration) / 1e9);
  }
}

void mpiDslashQcu(void *fermion_out, void *fermion_in, void *gauge,
                  QcuParam *param, int parity, QcuParam *grid) {
  const int lat_x = param->lattice_size[0] >> 1;
  const int lat_y = param->lattice_size[1];
  const int lat_z = param->lattice_size[2];
  const int lat_t = param->lattice_size[3];
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // mpi wilson dslash
    int node_size, node_rank, move_b, move_f;
    MPI_Comm_size(MPI_COMM_WORLD, &node_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &node_rank);
    const int grid_x = grid->lattice_size[0];
    const int grid_y = grid->lattice_size[1];
    const int grid_z = grid->lattice_size[2];
    const int grid_t = grid->lattice_size[3];
    const int grid_index_x = node_rank / grid_t / grid_z / grid_y;
    const int grid_index_y = node_rank / grid_t / grid_z % grid_y;
    const int grid_index_z = node_rank / grid_t % grid_z;
    const int grid_index_t = node_rank % grid_t;
    printf("node_size: %d \n", node_size); // debug
    printf("node_rank: %d \n", node_rank); // debug
    printf("grid:x-%d, y-%d, z-%d, t-%d \n", grid_x, grid_y, grid_z,
           grid_t); // debug
    printf("grid_index:x-%d, y-%d, z-%d, t-%d \n", grid_index_x, grid_index_y,
           grid_index_z, grid_index_t); // debug
    MPI_Request b_x_send_request, b_x_recv_request;
    MPI_Request f_x_send_request, f_x_recv_request;
    MPI_Request b_y_send_request, b_y_recv_request;
    MPI_Request f_y_send_request, f_y_recv_request;
    MPI_Request b_z_send_request, b_z_recv_request;
    MPI_Request f_z_send_request, f_z_recv_request;
    MPI_Request b_t_send_request, b_t_recv_request;
    MPI_Request f_t_send_request, f_t_recv_request;
    void *b_x_send_vec, *b_x_recv_vec;
    void *f_x_send_vec, *f_x_recv_vec;
    void *b_y_send_vec, *b_y_recv_vec;
    void *f_y_send_vec, *f_y_recv_vec;
    void *b_z_send_vec, *b_z_recv_vec;
    void *f_z_send_vec, *f_z_recv_vec;
    void *b_t_send_vec, *b_t_recv_vec;
    void *f_t_send_vec, *f_t_recv_vec;
    hipMallocManaged(&b_x_send_vec,
                      lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_x_send_vec,
                      lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_y_send_vec,
                      lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_y_send_vec,
                      lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_z_send_vec,
                      lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_z_send_vec,
                      lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_t_send_vec,
                      lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_t_send_vec,
                      lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_x_recv_vec,
                      lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_x_recv_vec,
                      lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_y_recv_vec,
                      lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_y_recv_vec,
                      lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_z_recv_vec,
                      lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_z_recv_vec,
                      lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_t_recv_vec,
                      lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_t_recv_vec,
                      lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    auto start = std::chrono::high_resolution_clock::now();
    // clean
    wilson_dslash_clear_dest<<<gridDim, blockDim>>>(fermion_out, lat_x, lat_y,
                                                    lat_z);
    // send x
    wilson_dslash_x_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_x_send_vec, f_x_send_vec);
    checkCudaErrors(hipDeviceSynchronize());
    if (grid_x != 1) {
      move_backward(move_b, grid_index_x, grid_x);
      move_forward(move_f, grid_index_x, grid_x);
      move_b = node_rank + move_b * grid_y * grid_z * grid_t;
      move_f = node_rank + move_f * grid_y * grid_z * grid_t;
      MPI_Irecv(b_x_recv_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_x_recv_request);
      MPI_Irecv(f_x_recv_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_x_recv_request);
      MPI_Isend(b_x_send_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_x_send_request);
      MPI_Isend(f_x_send_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_x_send_request);
      printf("######%d >-b_x-> %d######\n", node_rank, move_b); // debug
      printf("######%d >-f_x-> %d######\n", node_rank, move_f); // debug
    }
    // send y
    wilson_dslash_y_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_y_send_vec, f_y_send_vec);
    checkCudaErrors(hipDeviceSynchronize());
    if (grid_y != 1) {
      move_backward(move_b, grid_index_y, grid_y);
      move_forward(move_f, grid_index_y, grid_y);
      move_b = node_rank + move_b * grid_z * grid_t;
      move_f = node_rank + move_f * grid_z * grid_t;
      MPI_Irecv(b_y_recv_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_y_recv_request);
      MPI_Irecv(f_y_recv_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_y_recv_request);
      MPI_Isend(b_y_send_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_y_send_request);
      MPI_Isend(f_y_send_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_y_send_request);
      printf("######%d >-b_y-> %d######\n", node_rank, move_b); // debug
      printf("######%d >-f_y-> %d######\n", node_rank, move_f); // debug
    }
    // send z
    wilson_dslash_z_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_z_send_vec, f_z_send_vec);
    checkCudaErrors(hipDeviceSynchronize());
    if (grid_z != 1) {
      move_backward(move_b, grid_index_z, grid_z);
      move_forward(move_f, grid_index_z, grid_z);
      move_b = node_rank + move_b * grid_t;
      move_f = node_rank + move_f * grid_t;
      MPI_Irecv(b_z_recv_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_z_recv_request);
      MPI_Irecv(f_z_recv_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_z_recv_request);
      MPI_Isend(b_z_send_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_z_send_request);
      MPI_Isend(f_z_send_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_z_send_request);
      printf("######%d >-b_z-> %d######\n", node_rank, move_b); // debug
      printf("######%d >-f_z-> %d######\n", node_rank, move_f); // debug
    }
    // send t
    wilson_dslash_t_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_t_send_vec, f_t_send_vec);
    checkCudaErrors(hipDeviceSynchronize());
    if (grid_t != 1) {
      move_backward(move_b, grid_index_t, grid_t);
      move_forward(move_f, grid_index_t, grid_t);
      move_b = node_rank + move_b;
      move_f = node_rank + move_f;
      MPI_Irecv(b_t_recv_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_t_recv_request);
      MPI_Irecv(f_t_recv_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_t_recv_request);
      MPI_Isend(b_t_send_vec, 12, MPI_DOUBLE, move_b, move_b, MPI_COMM_WORLD,
                &b_t_send_request);
      MPI_Isend(f_t_send_vec, 12, MPI_DOUBLE, move_f, node_rank, MPI_COMM_WORLD,
                &f_t_send_request);
      printf("######%d >-b_t-> %d######\n", node_rank, move_b); // debug
      printf("######%d >-f_t-> %d######\n", node_rank, move_f); // debug
    }
    // recv x
    if (grid_x != 1) {
      MPI_Wait(&b_x_recv_request, MPI_STATUS_IGNORE);
      MPI_Wait(&f_x_recv_request, MPI_STATUS_IGNORE);
      wilson_dslash_x_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_x_recv_vec, f_x_recv_vec);
    } else {
      wilson_dslash_x_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_x, lat_y, lat_z, lat_t, parity, f_x_send_vec,
          b_x_send_vec); // debug
    }
    // recv y
    if (grid_y != 1) {
      MPI_Wait(&b_y_recv_request, MPI_STATUS_IGNORE);
      MPI_Wait(&f_y_recv_request, MPI_STATUS_IGNORE);
      wilson_dslash_y_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_y_recv_vec, f_y_recv_vec);
    } else {
      wilson_dslash_y_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  f_y_send_vec, b_y_send_vec);
    }
    // recv z
    if (grid_z != 1) {
      MPI_Wait(&b_z_recv_request, MPI_STATUS_IGNORE);
      MPI_Wait(&f_z_recv_request, MPI_STATUS_IGNORE);
      wilson_dslash_z_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_z_recv_vec, f_z_recv_vec);
    } else {
      wilson_dslash_z_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  f_z_send_vec, b_z_send_vec);
    }
    // recv t
    if (grid_t != 1) {
      MPI_Wait(&b_t_recv_request, MPI_STATUS_IGNORE);
      MPI_Wait(&f_t_recv_request, MPI_STATUS_IGNORE);
      wilson_dslash_t_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_t_recv_vec, f_t_recv_vec);
    } else {
      wilson_dslash_t_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  f_t_send_vec, b_t_send_vec);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    {
      checkCudaErrors(hipDeviceSynchronize());
      auto end = std::chrono::high_resolution_clock::now();
      auto duration =
          std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
              .count();
      err = hipGetLastError();
      checkCudaErrors(err);
      //printf("#######DEBUG####### \n"); // debug
      //print_tmp(b_x_send_vec, 6);       // debug
      //print_tmp(f_x_send_vec, 6);       // debug
      //print_tmp(b_y_send_vec, 6);       // debug
      //print_tmp(f_y_send_vec, 6);       // debug
      //print_tmp(b_z_send_vec, 6);       // debug
      //print_tmp(f_z_send_vec, 6);       // debug
      //print_tmp(b_t_send_vec, 6);       // debug
      //print_tmp(f_t_send_vec, 6);       // debug
      //printf("#######DEBUG####### \n"); // debug
      printf(
          "mpi wilson dslash total time: (without malloc free memcpy) :%.9lf "
          "sec\n",
          double(duration) / 1e9);
    }
    {
      // free
      checkCudaErrors(hipFree(b_x_send_vec));
      checkCudaErrors(hipFree(f_x_send_vec));
      checkCudaErrors(hipFree(b_y_send_vec));
      checkCudaErrors(hipFree(f_y_send_vec));
      checkCudaErrors(hipFree(b_z_send_vec));
      checkCudaErrors(hipFree(f_z_send_vec));
      checkCudaErrors(hipFree(b_t_send_vec));
      checkCudaErrors(hipFree(f_t_send_vec));
      checkCudaErrors(hipFree(b_x_recv_vec));
      checkCudaErrors(hipFree(f_x_recv_vec));
      checkCudaErrors(hipFree(b_y_recv_vec));
      checkCudaErrors(hipFree(f_y_recv_vec));
      checkCudaErrors(hipFree(b_z_recv_vec));
      checkCudaErrors(hipFree(f_z_recv_vec));
      checkCudaErrors(hipFree(b_t_recv_vec));
      checkCudaErrors(hipFree(f_t_recv_vec));
    }
  }
}
