#include "../../include/qcu.h"
#include "../../include/qcu_cuda.h"
#include <chrono>
#include <cstdio>

void dslashQcu(void *fermion_out, void *fermion_in, void *gauge,
               QcuParam *param, int parity) {
  const int lat_x = param->lattice_size[0] >> 1;
  const int lat_y = param->lattice_size[1];
  const int lat_z = param->lattice_size[2];
  const int lat_t = param->lattice_size[3];
  void *clover;
  checkCudaErrors(hipMalloc(&clover, (lat_t * lat_z * lat_y * lat_x * 144) *
                                          sizeof(LatticeComplex)));
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // wilson dslash
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    wilson_dslash<<<gridDim, blockDim>>>(gauge, fermion_in, fermion_out, lat_x,
                                         lat_y, lat_z, lat_t, parity);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf(
        "wilson dslash total time: (without malloc free memcpy) : %.9lf sec\n",
        double(duration) / 1e9);
  }
  {
    // make clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    make_clover<<<gridDim, blockDim>>>(gauge, clover, lat_x, lat_y, lat_z,
                                       lat_t, parity);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("make clover total time: (without malloc free memcpy) :%.9lf sec\n ",
           double(duration) / 1e9);
  }
  {
    // inverse clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    inverse_clover<<<gridDim, blockDim>>>(clover, lat_x, lat_y, lat_z);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf(
        "inverse clover total time: (without malloc free memcpy) :%.9lf sec\n ",
        double(duration) / 1e9);
  }
  {
    // give clover
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    give_clover<<<gridDim, blockDim>>>(clover, fermion_out, lat_x, lat_y,
                                       lat_z);
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("give clover total time: (without malloc free memcpy) :%.9lf sec\n ",
           double(duration) / 1e9);
  }
  {
    // free
    checkCudaErrors(hipFree(clover));
  }
}

void mpiDslashQcu(void *fermion_out, void *fermion_in, void *gauge,
                  QcuParam *param, int parity, QcuParam *grid) {
  int node_size, node_rank, move;
  MPI_Comm_size(MPI_COMM_WORLD, &node_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &node_rank);
  const int grid_x = grid->lattice_size[0];
  const int grid_y = grid->lattice_size[1];
  const int grid_z = grid->lattice_size[2];
  const int grid_t = grid->lattice_size[3];
  const int lat_x = param->lattice_size[0] >> 1 / grid_x;
  const int lat_y = param->lattice_size[1] / grid_y;
  const int lat_z = param->lattice_size[2] / grid_z;
  const int lat_t = param->lattice_size[3] / grid_t;
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // mpi wilson dslash
    const int grid_index_x = node_rank / grid_t / grid_z / grid_y;
    const int grid_index_y = node_rank / grid_t / grid_z % grid_y;
    const int grid_index_z = node_rank / grid_t % grid_z;
    const int grid_index_t = node_rank % grid_t;
    printf("node_size: %d \n", node_size);
    printf("node_rank: %d \n", node_rank);
    printf("grid:x-%d, y-%d, z-%d, t-%d \n", grid_x, grid_y, grid_z, grid_t);
    printf("grid_index:x-%d, y-%d, z-%d, t-%d \n", grid_index_x, grid_index_y,
           grid_index_z, grid_index_t);
    MPI_Request b_x_send_request;
    MPI_Request f_x_send_request;
    MPI_Request b_y_send_request;
    MPI_Request f_y_send_request;
    MPI_Request b_z_send_request;
    MPI_Request f_z_send_request;
    MPI_Request b_t_send_request;
    MPI_Request f_t_send_request;
    void *b_x_send_vec =
        (void *)malloc(lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    void *f_x_send_vec =
        (void *)malloc(lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    void *b_y_send_vec =
        (void *)malloc(lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    void *f_y_send_vec =
        (void *)malloc(lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    void *b_z_send_vec =
        (void *)malloc(lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *f_z_send_vec =
        (void *)malloc(lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *b_t_send_vec =
        (void *)malloc(lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *f_t_send_vec =
        (void *)malloc(lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    MPI_Request b_x_recv_request;
    MPI_Request f_x_recv_request;
    MPI_Request b_y_recv_request;
    MPI_Request f_y_recv_request;
    MPI_Request b_z_recv_request;
    MPI_Request f_z_recv_request;
    MPI_Request b_t_recv_request;
    MPI_Request f_t_recv_request;
    void *b_x_recv_vec =
        (void *)malloc(lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    void *f_x_recv_vec =
        (void *)malloc(lat_t * lat_z * lat_y * 6 * sizeof(LatticeComplex));
    void *b_y_recv_vec =
        (void *)malloc(lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    void *f_y_recv_vec =
        (void *)malloc(lat_t * lat_z * lat_x * 6 * sizeof(LatticeComplex));
    void *b_z_recv_vec =
        (void *)malloc(lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *f_z_recv_vec =
        (void *)malloc(lat_t * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *b_t_recv_vec =
        (void *)malloc(lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    void *f_t_recv_vec =
        (void *)malloc(lat_z * lat_y * lat_x * 6 * sizeof(LatticeComplex));
    checkCudaErrors(hipDeviceSynchronize());
    auto start = std::chrono::high_resolution_clock::now();
    wilson_dslash_clear_dest<<<gridDim, blockDim>>>(fermion_out, lat_x, lat_y,
                                                    lat_z);
    // send x
    wilson_dslash_x_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_x_send_vec, f_x_send_vec);
    if (grid_x != 1) {
      move_backward(move, grid_index_x, grid_x);
      move = node_rank + move * grid_y * grid_z * grid_t;
      MPI_Isend(b_x_send_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_x_send_request);
      printf("######%d --> %d######\n", node_rank, move);
      move_forward(move, grid_index_x, grid_x);
      move = node_rank + move * grid_y * grid_z * grid_t;
      MPI_Isend(f_x_send_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_x_send_request);
      printf("######%d --> %d######\n", node_rank, move);
    }
    // send y
    wilson_dslash_y_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_y_send_vec, f_y_send_vec);
    if (grid_y != 1) {
      move_backward(move, grid_index_y, grid_y);
      move = node_rank + move * grid_z * grid_t;
      MPI_Isend(b_y_send_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_y_send_request);
      printf("######%d --> %d######\n", node_rank, move);
      move_forward(move, grid_index_y, grid_y);
      move = node_rank + move * grid_z * grid_t;
      MPI_Isend(f_y_send_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_y_send_request);
      printf("######%d --> %d######\n", node_rank, move);
    }
    // send z
    wilson_dslash_z_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_z_send_vec, f_z_send_vec);
    if (grid_z != 1) {
      move_backward(move, grid_index_z, grid_z);
      move = node_rank + move * grid_t;
      MPI_Isend(b_z_send_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_z_send_request);
      printf("######%d --> %d######\n", node_rank, move);
      move_forward(move, grid_index_z, grid_z);
      move = node_rank + move * grid_t;
      MPI_Isend(f_z_send_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_z_send_request);
      printf("######%d --> %d######\n", node_rank, move);
    }
    // send t
    wilson_dslash_t_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_x, lat_y, lat_z, lat_t, parity,
        b_t_send_vec, f_t_send_vec);
    if (grid_t != 1) {
      move_backward(move, grid_index_t, grid_t);
      move = node_rank + move;
      MPI_Isend(b_t_send_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_t_send_request);
      printf("######%d --> %d######\n", node_rank, move);
      move_forward(move, grid_index_t, grid_t);
      move = node_rank + move;
      MPI_Isend(f_t_send_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_t_send_request);
      printf("######%d --> %d######\n", node_rank, move);
    }
    printf("#######DEBUG####### \n");
    print_tmp(b_x_send_vec, 100);
    print_tmp(f_x_send_vec, 100);
    print_tmp(b_y_send_vec, 100);
    print_tmp(f_y_send_vec, 100);
    print_tmp(b_z_send_vec, 100);
    print_tmp(f_z_send_vec, 100);
    print_tmp(b_t_send_vec, 100);
    print_tmp(f_t_send_vec, 100);
    printf("#######DEBUG####### \n");
    // recv x
    if (grid_x != 1) {
      move_backward(move, grid_index_x, grid_x);
      move = node_rank + move * grid_y * grid_z * grid_t;
      MPI_Wait(&b_x_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(b_x_recv_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_x_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      move_forward(move, grid_index_x, grid_x);
      move = node_rank + move * grid_y * grid_z * grid_t;
      MPI_Wait(&f_x_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(f_x_recv_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_x_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      wilson_dslash_x_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_x_recv_vec, f_x_recv_vec);
    }
    // recv y
    if (grid_y != 1) {
      move_backward(move, grid_index_y, grid_y);
      move = node_rank + move * grid_z * grid_t;
      MPI_Wait(&b_y_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(b_y_recv_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_y_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      move_forward(move, grid_index_y, grid_y);
      move = node_rank + move * grid_z * grid_t;
      MPI_Wait(&f_y_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(f_y_recv_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_y_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      wilson_dslash_y_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_y_recv_vec, f_y_recv_vec);
    }
    // recv z
    if (grid_z != 1) {
      move_backward(move, grid_index_z, grid_z);
      move = node_rank + move * grid_t;
      MPI_Wait(&b_z_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(b_z_recv_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_z_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      move_forward(move, grid_index_z, grid_z);
      move = node_rank + move * grid_t;
      MPI_Wait(&f_z_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(f_z_recv_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_z_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      wilson_dslash_z_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_z_recv_vec, f_z_recv_vec);
    }
    // recv t
    if (grid_t != 1) {
      move_backward(move, grid_index_t, grid_t);
      move = node_rank + move;
      MPI_Wait(&b_t_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(b_t_recv_vec, 12, MPI_DOUBLE, move, move, MPI_COMM_WORLD,
                &b_t_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      move_forward(move, grid_index_t, grid_t);
      move = node_rank + move;
      MPI_Wait(&f_t_send_request, MPI_STATUS_IGNORE);
      MPI_Irecv(f_t_recv_vec, 12, MPI_DOUBLE, move, node_rank, MPI_COMM_WORLD,
                &f_t_recv_request);
      printf("######%d <-- %d######\n", node_rank, move);
      wilson_dslash_t_recv<<<gridDim, blockDim>>>(gauge, fermion_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_t_recv_vec, f_t_recv_vec);
    }
    err = hipGetLastError();
    checkCudaErrors(err);
    checkCudaErrors(hipDeviceSynchronize());
    MPI_Barrier(MPI_COMM_WORLD);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    printf("mpi wilson dslash total time: (without malloc free memcpy) :%.9lf "
           "sec\n",
           double(duration) / 1e9);
    {
      // free
      free(b_x_send_vec);
      free(f_x_send_vec);
      free(b_y_send_vec);
      free(f_y_send_vec);
      free(b_z_send_vec);
      free(f_z_send_vec);
      free(b_t_send_vec);
      free(f_t_send_vec);
      free(b_x_recv_vec);
      free(f_x_recv_vec);
      free(b_y_recv_vec);
      free(f_y_recv_vec);
      free(b_z_recv_vec);
      free(f_z_recv_vec);
      free(b_t_recv_vec);
      free(f_t_recv_vec);
    }
  }
}
