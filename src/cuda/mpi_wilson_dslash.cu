#include "hip/hip_runtime.h"
#pragma optimize(5)
#include "../../include/qcu_cuda.h"

__global__ void mpi_wilson_dslash_f_x(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_f_y(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_f_z(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_f_t(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_b_x(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_b_y(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_b_z(void *U, void *src) {
}
__global__ void mpi_wilson_dslash_b_t(void *U, void *src) {
}

__global__ void mpi_wilson_dslash(void *device_U, void *device_src,
                              void *device_dest, int device_lat_x,
                              const int device_lat_y, const int device_lat_z,
                              const int device_lat_t, const int device_parity, int device_grid_x,
                              const int device_grid_y, const int device_grid_z,
                              const int device_grid_t) {
  int node_size, node_rank;
  MPI_Comm_size(MPI_COMM_WORLD, &node_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &node_rank);                        
  register int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  const int lat_xcc = lat_x * 9;
  const int lat_yxcc = lat_y * lat_xcc;
  const int lat_zyxcc = lat_z * lat_yxcc;
  const int lat_tzyxcc = lat_t * lat_zyxcc;
  const int lat_xsc = lat_x * 12;
  const int lat_yxsc = lat_y * lat_xsc;
  const int lat_zyxsc = lat_z * lat_yxsc;
  register int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  const int eo = (y + z + t) & 0x01; // (y+z+t)%2
  register LatticeComplex I(0.0, 1.0);
  register LatticeComplex zero(0.0, 0.0);
  register LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) + t * lat_zyxcc +
       z * lat_yxcc + y * lat_xcc + x * 9);
  register LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  register LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  register LatticeComplex *tmp_U;
  register LatticeComplex *tmp_src;
  register LatticeComplex tmp0(0.0, 0.0);
  register LatticeComplex tmp1(0.0, 0.0);
  register LatticeComplex U[9];
  register LatticeComplex src[12];
  register LatticeComplex dest[12];
  // just wilson(Sum part)
  give_value(dest, zero, 12);
  {
    // x-1
    move_backward_x(move, x, lat_x, eo, parity);
    tmp_U = (origin_U + move * 9 + (1 - parity) * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * 12);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] + src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
        tmp1 += (src[c1 + 3] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] -= tmp1 * I;
      dest[c0 + 9] -= tmp0 * I;
    }
  }
  {
    // x+1
    move_forward_x(move, x, lat_x, eo, parity);
    tmp_U = (origin_U + parity * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * 12);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] - src[c1 + 9] * I) * U[c0 * 3 + c1];
        tmp1 += (src[c1 + 3] - src[c1 + 6] * I) * U[c0 * 3 + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] += tmp1 * I;
      dest[c0 + 9] += tmp0 * I;
    }
  }
  {
    // y-1
    move_backward(move, y, lat_y);
    tmp_U = (origin_U + move * lat_xcc + lat_tzyxcc * 2 +
             (1 - parity) * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_xsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] - src[c1 + 9]) * U[c1 * 3 + c0].conj();
        tmp1 += (src[c1 + 3] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] += tmp1;
      dest[c0 + 9] -= tmp0;
    }
  }
  {
    // y+1
    move_forward(move, y, lat_y);
    tmp_U = (origin_U + lat_tzyxcc * 2 + parity * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_xsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] + src[c1 + 9]) * U[c0 * 3 + c1];
        tmp1 += (src[c1 + 3] - src[c1 + 6]) * U[c0 * 3 + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] -= tmp1;
      dest[c0 + 9] += tmp0;
    }
  }
  {
    // z-1
    move_backward(move, z, lat_z);
    tmp_U = (origin_U + move * lat_yxcc + lat_tzyxcc * 4 +
             (1 - parity) * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_yxsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
        tmp1 += (src[c1 + 3] - src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] -= tmp0 * I;
      dest[c0 + 9] += tmp1 * I;
    }
  }
  {
    // z+1
    move_forward(move, z, lat_z);
    tmp_U = (origin_U + lat_tzyxcc * 4 + parity * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_yxsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] - src[c1 + 6] * I) * U[c0 * 3 + c1];
        tmp1 += (src[c1 + 3] + src[c1 + 9] * I) * U[c0 * 3 + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] += tmp0 * I;
      dest[c0 + 9] -= tmp1 * I;
    }
  }
  {
    // t-1
    move_backward(move, t, lat_t);
    tmp_U = (origin_U + move * lat_zyxcc + lat_tzyxcc * 6 +
             (1 - parity) * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_zyxsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
        tmp1 += (src[c1 + 3] + src[c1 + 9]) * U[c1 * 3 + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] += tmp0;
      dest[c0 + 9] += tmp1;
    }
  }
  {
    // t+1
    move_forward(move, t, lat_t);
    tmp_U = (origin_U + lat_tzyxcc * 6 + parity * lat_tzyxcc);
    give_u(U, tmp_U);
    tmp_src = (origin_src + move * lat_zyxsc);
    give_ptr(src, tmp_src, 12);
  }
  {
    for (int c0 = 0; c0 < 3; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < 3; c1++) {
        tmp0 += (src[c1] - src[c1 + 6]) * U[c0 * 3 + c1];
        tmp1 += (src[c1 + 3] - src[c1 + 9]) * U[c0 * 3 + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + 3] += tmp1;
      dest[c0 + 6] -= tmp0;
      dest[c0 + 9] -= tmp1;
    }
  }
  give_ptr(origin_dest, dest, 12);
}
