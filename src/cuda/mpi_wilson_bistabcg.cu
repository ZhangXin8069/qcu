#include <iostream>
#pragma optimize(5)
#include "../../include/qcu.h"
#ifdef MPI_WILSON_BISTABCG
// #define DEBUG_MPI_WILSON_CG
void mpiBistabCgQcu(void *gauge, QcuParam *param, QcuParam *grid) {
  // define for mpi_wilson_dslash
  int lat_1dim[DIM];
  int lat_3dim[DIM];
  int lat_4dim;
  give_dims(param, lat_1dim, lat_3dim, lat_4dim);
  int lat_3dim6[DIM];
  int lat_3dim12[DIM];
  for (int i = 0; i < DIM; i++) {
    lat_3dim6[i] = lat_3dim[i] * 6;
    lat_3dim12[i] = lat_3dim6[i] * 2;
  }
  hipError_t err;
  dim3 gridDim(lat_4dim / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  int node_rank;
  int move[BF];
  int grid_1dim[DIM];
  int grid_index_1dim[DIM];
  give_grid(grid, node_rank, grid_1dim, grid_index_1dim);
  MPI_Request send_request[WARDS];
  MPI_Request recv_request[WARDS];
  void *host_send_vec[WARDS];
  void *host_recv_vec[WARDS];
  void *device_send_vec[WARDS];
  void *device_recv_vec[WARDS];
  malloc_vec(lat_3dim6, device_send_vec, device_recv_vec, host_send_vec,
             host_recv_vec);
  // define end
  // define for mpi_wilson_cg
  int lat_4dim12 = lat_4dim * 12;
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex one(1.0, 0.0);
  LatticeComplex r_norm2(0.0, 0.0);
  const int MAX_ITER(1e3); // 300++?
  const double TOL(1e-6);
  LatticeComplex rho_prev(1.0, 0.0);
  LatticeComplex rho(0.0, 0.0);
  LatticeComplex alpha(1.0, 0.0);
  LatticeComplex omega(1.0, 0.0);
  LatticeComplex beta(0.0, 0.0);
  double kappa = 0.125;
  LatticeComplex tmp(0.0, 0.0);
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex local_result(0.0, 0.0);
  void *ans_e, *ans_o, *x_e, *x_o, *b_e, *b_o, *b__o, *r, *r_tilde, *p, *v, *s,
      *t, *device_latt_tmp0, *device_latt_tmp1;
  hipMalloc(&ans_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&ans_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&x_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&x_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b_e, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b_o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&b__o, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&r, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&r_tilde, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&p, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&v, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&s, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&t, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&device_latt_tmp0, lat_4dim12 * sizeof(LatticeComplex));
  hipMalloc(&device_latt_tmp1, lat_4dim12 * sizeof(LatticeComplex));
  LatticeComplex *host_latt_tmp0 =
      (LatticeComplex *)malloc(lat_4dim12 * sizeof(LatticeComplex));
  LatticeComplex *host_latt_tmp1 =
      (LatticeComplex *)malloc(lat_4dim12 * sizeof(LatticeComplex));
  // give ans first
  give_random_value<<<gridDim, blockDim>>>(ans_e, node_rank);
  give_random_value<<<gridDim, blockDim>>>(ans_o, node_rank);
  // give x_o, b_e, b_o ,b__o, r, r_tilde, p, v, s, t
  give_random_value<<<gridDim, blockDim>>>(x_o, node_rank);
  give_custom_value<<<gridDim, blockDim>>>(b_e, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(b_o, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(b__o, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(r, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(r_tilde, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(p, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(v, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(s, 0.0, 0.0);
  give_custom_value<<<gridDim, blockDim>>>(t, 0.0, 0.0);
  // give b'_o(b__0)
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp0, 0.0, 0.0);
  mpi_dslash_eo(device_latt_tmp0, ans_o, node_rank, gridDim, blockDim, gauge,
                lat_1dim, lat_3dim12, grid_1dim, grid_index_1dim, move,
                send_request, recv_request, device_send_vec, device_recv_vec,
                host_send_vec, host_recv_vec, zero);
  wilson_bistabcg_give_b_e<<<gridDim, blockDim>>>(void *b_e, void *ans_e,
                                      void *device_latt_tmp0, double kappa);
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp1, 0.0, 0.0);
  mpi_dslash_oe(device_latt_tmp1, ans_e, node_rank, gridDim, blockDim, gauge,
                lat_1dim, lat_3dim12, grid_1dim, grid_index_1dim, move,
                send_request, recv_request, device_send_vec, device_recv_vec,
                host_send_vec, host_recv_vec, zero);
  wilson_bistabcg_give_b_o<<<gridDim, blockDim>>>(void *b_o, void *ans_o,
                                      void *device_latt_tmp1, double kappa);
  give_custom_value<<<gridDim, blockDim>>>(device_latt_tmp0, 0.0, 0.0);

  auto start = std::chrono::high_resolution_clock::now();
  checkCudaErrors(hipDeviceSynchronize());
  hipMemcpy(host_latt_tmp0, device_latt_tmp0,
             lat_4dim12 * sizeof(LatticeComplex), hipMemcpyDeviceToHost);
  checkCudaErrors(hipDeviceSynchronize());
  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  err = hipGetLastError();
  checkCudaErrors(err);
  printf("mpi wilson bistabcg total time: (without malloc free "
         "memcpy) :%.9lf "
         "sec\n",
         double(duration) / 1e9);
  printf("[0]:%f,%f;[-1]:%f,%f;\n", host_latt_tmp0[0].real,
         host_latt_tmp0[0].imag, host_latt_tmp0[lat_4dim12 - 1].real,
         host_latt_tmp0[lat_4dim12 - 1].imag);
  // free
  free_vec(device_send_vec, device_recv_vec, host_send_vec, host_recv_vec);
  hipFree(ans_e);
  hipFree(ans_o);
  hipFree(x_o);
  hipFree(b__o);
  hipFree(r);
  hipFree(r_tilde);
  hipFree(p);
  hipFree(v);
  hipFree(s);
  hipFree(t);
  hipFree(device_latt_tmp0);
  hipFree(device_latt_tmp1);
  free(host_latt_tmp0);
  free(host_latt_tmp1);
}
#endif