#include <iostream>
#pragma optimize(5)
#include "../../include/qcu.h"

#ifdef MPI_WILSON_BISTABCG
void mpiCgQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param,
              int parity, QcuParam *grid) {
  const int lat_x = param->lattice_size[0] >> 1;
  const int lat_y = param->lattice_size[1];
  const int lat_z = param->lattice_size[2];
  const int lat_t = param->lattice_size[3];
  const int lat_yzt6 = lat_y * lat_z * lat_t * 6;
  const int lat_xzt6 = lat_x * lat_z * lat_t * 6;
  const int lat_xyt6 = lat_x * lat_y * lat_t * 6;
  const int lat_xyz6 = lat_x * lat_y * lat_z * 6;
  const int lat_yzt12 = lat_yzt6 * 2;
  const int lat_xzt12 = lat_xzt6 * 2;
  const int lat_xyt12 = lat_xyt6 * 2;
  const int lat_xyz12 = lat_xyz6 * 2;
  const int lat_xyzt12 = lat_xyz6 * lat_t * 2;
  hipError_t err;
  dim3 gridDim(lat_x * lat_y * lat_z * lat_t / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // mpi wilson cg
    int node_size, node_rank, move_b, move_f;
    MPI_Comm_size(MPI_COMM_WORLD, &node_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &node_rank);
    const int grid_x = grid->lattice_size[0];
    const int grid_y = grid->lattice_size[1];
    const int grid_z = grid->lattice_size[2];
    const int grid_t = grid->lattice_size[3];
    const int grid_index_x = node_rank / grid_t / grid_z / grid_y;
    const int grid_index_y = node_rank / grid_t / grid_z % grid_y;
    const int grid_index_z = node_rank / grid_t % grid_z;
    const int grid_index_t = node_rank % grid_t;
    MPI_Request b_x_send_request, b_x_recv_request;
    MPI_Request f_x_send_request, f_x_recv_request;
    MPI_Request b_y_send_request, b_y_recv_request;
    MPI_Request f_y_send_request, f_y_recv_request;
    MPI_Request b_z_send_request, b_z_recv_request;
    MPI_Request f_z_send_request, f_z_recv_request;
    MPI_Request b_t_send_request, b_t_recv_request;
    MPI_Request f_t_send_request, f_t_recv_request;
    void *b_x_send_vec, *b_x_recv_vec;
    void *f_x_send_vec, *f_x_recv_vec;
    void *b_y_send_vec, *b_y_recv_vec;
    void *f_y_send_vec, *f_y_recv_vec;
    void *b_z_send_vec, *b_z_recv_vec;
    void *f_z_send_vec, *f_z_recv_vec;
    void *b_t_send_vec, *b_t_recv_vec;
    void *f_t_send_vec, *f_t_recv_vec;
    hipMallocManaged(&b_x_send_vec, lat_yzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_x_send_vec, lat_yzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_y_send_vec, lat_xzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_y_send_vec, lat_xzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_z_send_vec, lat_xyt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_z_send_vec, lat_xyt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_t_send_vec, lat_xyz6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_t_send_vec, lat_xyz6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_x_recv_vec, lat_yzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_x_recv_vec, lat_yzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_y_recv_vec, lat_xzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_y_recv_vec, lat_xzt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_z_recv_vec, lat_xyt6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_z_recv_vec, lat_xyt6 * sizeof(LatticeComplex));
    hipMallocManaged(&b_t_recv_vec, lat_xyz6 * sizeof(LatticeComplex));
    hipMallocManaged(&f_t_recv_vec, lat_xyz6 * sizeof(LatticeComplex));
    LatticeComplex *cg_in, *cg_out, *x, *b, *r, *r_tilde, *p, *v, *s, *t;
    hipMallocManaged(&x, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&b, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&r, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&r_tilde, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&p, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&v, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&s, lat_xyzt12 * sizeof(LatticeComplex));
    hipMallocManaged(&t, lat_xyzt12 * sizeof(LatticeComplex));
    LatticeComplex zero(0.0, 0.0);
    LatticeComplex one(1.0, 0.0);
    const int MAX_ITER(1e2);
    const double TOL(1e-6);
    LatticeComplex rho_prev(1.0, 0.0);
    LatticeComplex rho(0.0, 0.0);
    LatticeComplex alpha(1.0, 0.0);
    LatticeComplex omega(1.0, 0.0);
    LatticeComplex beta(0.0, 0.0);
    LatticeComplex tmp(0.0, 0.0);
    LatticeComplex tmp0(0.0, 0.0);
    LatticeComplex tmp1(0.0, 0.0);
    LatticeComplex r_norm2(0.0, 0.0);
    LatticeComplex local_result = 0;
    // double Kappa = 0.125;
    double Kappa = -7.0;
    // above define for mpi_wilson_dslash and mpi_wilson_cg
    auto start = std::chrono::high_resolution_clock::now();
    give_rand(x, lat_xyzt12); // rand x
    // give_rand(b, lat_xyzt12); // rand source
    // give_value(b, one, 1);    // point source
    cg_in = x;
    cg_out = r;
    // mpi_wilson_dslash
    {
      // clean
      wilson_dslash_clear_dest<<<gridDim, blockDim>>>(cg_out, lat_x, lat_y,
                                                      lat_z);
      // send x
      wilson_dslash_x_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_x_send_vec, f_x_send_vec);
      if (grid_x != 1) {
        checkCudaErrors(hipDeviceSynchronize());
        move_backward(move_b, grid_index_x, grid_x);
        move_forward(move_f, grid_index_x, grid_x);
        move_b = node_rank + move_b * grid_y * grid_z * grid_t;
        move_f = node_rank + move_f * grid_y * grid_z * grid_t;
        MPI_Irecv(b_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_b, 1,
                  MPI_COMM_WORLD, &b_x_recv_request);
        MPI_Irecv(f_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_f, 0,
                  MPI_COMM_WORLD, &f_x_recv_request);
        MPI_Isend(b_x_send_vec, lat_yzt12, MPI_DOUBLE, move_b, 0,
                  MPI_COMM_WORLD, &b_x_send_request);
        MPI_Isend(f_x_send_vec, lat_yzt12, MPI_DOUBLE, move_f, 1,
                  MPI_COMM_WORLD, &f_x_send_request);
      }
      // send y
      wilson_dslash_y_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_y_send_vec, f_y_send_vec);
      if (grid_y != 1) {
        checkCudaErrors(hipDeviceSynchronize());
        move_backward(move_b, grid_index_y, grid_y);
        move_forward(move_f, grid_index_y, grid_y);
        move_b = node_rank + move_b * grid_z * grid_t;
        move_f = node_rank + move_f * grid_z * grid_t;
        MPI_Irecv(b_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_b, 3,
                  MPI_COMM_WORLD, &b_y_recv_request);
        MPI_Irecv(f_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_f, 2,
                  MPI_COMM_WORLD, &f_y_recv_request);
        MPI_Isend(b_y_send_vec, lat_xzt12, MPI_DOUBLE, move_b, 2,
                  MPI_COMM_WORLD, &b_y_send_request);
        MPI_Isend(f_y_send_vec, lat_xzt12, MPI_DOUBLE, move_f, 3,
                  MPI_COMM_WORLD, &f_y_send_request);
      }
      // send z
      wilson_dslash_z_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_z_send_vec, f_z_send_vec);
      if (grid_z != 1) {
        checkCudaErrors(hipDeviceSynchronize());
        move_backward(move_b, grid_index_z, grid_z);
        move_forward(move_f, grid_index_z, grid_z);
        move_b = node_rank + move_b * grid_t;
        move_f = node_rank + move_f * grid_t;
        MPI_Irecv(b_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_b, 5,
                  MPI_COMM_WORLD, &b_z_recv_request);
        MPI_Irecv(f_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_f, 4,
                  MPI_COMM_WORLD, &f_z_recv_request);
        MPI_Isend(b_z_send_vec, lat_xyt12, MPI_DOUBLE, move_b, 4,
                  MPI_COMM_WORLD, &b_z_send_request);
        MPI_Isend(f_z_send_vec, lat_xyt12, MPI_DOUBLE, move_f, 5,
                  MPI_COMM_WORLD, &f_z_send_request);
      }
      // send t
      wilson_dslash_t_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                  lat_y, lat_z, lat_t, parity,
                                                  b_t_send_vec, f_t_send_vec);
      if (grid_t != 1) {
        checkCudaErrors(hipDeviceSynchronize());
        move_backward(move_b, grid_index_t, grid_t);
        move_forward(move_f, grid_index_t, grid_t);
        move_b = node_rank + move_b;
        move_f = node_rank + move_f;
        MPI_Irecv(b_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_b, 7,
                  MPI_COMM_WORLD, &b_t_recv_request);
        MPI_Irecv(f_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_f, 6,
                  MPI_COMM_WORLD, &f_t_recv_request);
        MPI_Isend(b_t_send_vec, lat_xyz12, MPI_DOUBLE, move_b, 6,
                  MPI_COMM_WORLD, &b_t_send_request);
        MPI_Isend(f_t_send_vec, lat_xyz12, MPI_DOUBLE, move_f, 7,
                  MPI_COMM_WORLD, &f_t_send_request);
      }
      // recv x
      if (grid_x != 1) {
        MPI_Wait(&b_x_recv_request, MPI_STATUS_IGNORE);
        MPI_Wait(&f_x_recv_request, MPI_STATUS_IGNORE);
        wilson_dslash_x_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    b_x_recv_vec, f_x_recv_vec);
      } else {
        checkCudaErrors(hipDeviceSynchronize());
        wilson_dslash_x_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    f_x_send_vec, b_x_send_vec);
      }
      // recv y
      if (grid_y != 1) {
        MPI_Wait(&b_y_recv_request, MPI_STATUS_IGNORE);
        MPI_Wait(&f_y_recv_request, MPI_STATUS_IGNORE);
        wilson_dslash_y_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    b_y_recv_vec, f_y_recv_vec);
      } else {
        checkCudaErrors(hipDeviceSynchronize());
        wilson_dslash_y_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    f_y_send_vec, b_y_send_vec);
      }
      // recv z
      if (grid_z != 1) {
        MPI_Wait(&b_z_recv_request, MPI_STATUS_IGNORE);
        MPI_Wait(&f_z_recv_request, MPI_STATUS_IGNORE);
        wilson_dslash_z_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    b_z_recv_vec, f_z_recv_vec);
      } else {
        checkCudaErrors(hipDeviceSynchronize());
        wilson_dslash_z_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    f_z_send_vec, b_z_send_vec);
      }
      // recv t
      if (grid_t != 1) {
        MPI_Wait(&b_t_recv_request, MPI_STATUS_IGNORE);
        MPI_Wait(&f_t_recv_request, MPI_STATUS_IGNORE);
        wilson_dslash_t_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    b_t_recv_vec, f_t_recv_vec);
      } else {
        checkCudaErrors(hipDeviceSynchronize());
        wilson_dslash_t_recv<<<gridDim, blockDim>>>(gauge, cg_out, lat_x, lat_y,
                                                    lat_z, lat_t, parity,
                                                    f_t_send_vec, b_t_send_vec);
      }
      MPI_Barrier(MPI_COMM_WORLD);
    }
    // kappa
    {
      for (int i = 0; i < lat_xyzt12; i++) {
        cg_out[i] = cg_out[i] - cg_in[i] * Kappa;
      }
    }
    for (int i = 0; i < lat_xyzt12; i++) {
      r[i] = b[i] - r[i];
      r_tilde[i] = r[i];
    }
    for (int loop = 0; loop < MAX_ITER; loop++) {
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          local_result = r_tilde[i].conj() * r[i];
        }
        MPI_Allreduce(&local_result, &rho, 2, MPI_DOUBLE, MPI_SUM,
                      MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
      }
#ifdef DEBUG
      std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
                << "##rho:" << rho.real << std::endl;
#endif
      beta = (rho / rho_prev) * (alpha / omega);
#ifdef DEBUG
      std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
                << "##beta:" << beta.real << std::endl;
#endif
      for (int i = 0; i < lat_xyzt12; i++) {
        p[i] = r[i] + (p[i] - v[i] * omega) * beta;
      }
      // v = A * p;
      cg_in = p;
      cg_out = v;
      // mpi_wilson_dslash
      {
        // clean
        wilson_dslash_clear_dest<<<gridDim, blockDim>>>(cg_out, lat_x, lat_y,
                                                        lat_z);
        // send x
        wilson_dslash_x_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_x_send_vec, f_x_send_vec);
        if (grid_x != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_x, grid_x);
          move_forward(move_f, grid_index_x, grid_x);
          move_b = node_rank + move_b * grid_y * grid_z * grid_t;
          move_f = node_rank + move_f * grid_y * grid_z * grid_t;
          MPI_Irecv(b_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_b, 1,
                    MPI_COMM_WORLD, &b_x_recv_request);
          MPI_Irecv(f_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_f, 0,
                    MPI_COMM_WORLD, &f_x_recv_request);
          MPI_Isend(b_x_send_vec, lat_yzt12, MPI_DOUBLE, move_b, 0,
                    MPI_COMM_WORLD, &b_x_send_request);
          MPI_Isend(f_x_send_vec, lat_yzt12, MPI_DOUBLE, move_f, 1,
                    MPI_COMM_WORLD, &f_x_send_request);
        }
        // send y
        wilson_dslash_y_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_y_send_vec, f_y_send_vec);
        if (grid_y != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_y, grid_y);
          move_forward(move_f, grid_index_y, grid_y);
          move_b = node_rank + move_b * grid_z * grid_t;
          move_f = node_rank + move_f * grid_z * grid_t;
          MPI_Irecv(b_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_b, 3,
                    MPI_COMM_WORLD, &b_y_recv_request);
          MPI_Irecv(f_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_f, 2,
                    MPI_COMM_WORLD, &f_y_recv_request);
          MPI_Isend(b_y_send_vec, lat_xzt12, MPI_DOUBLE, move_b, 2,
                    MPI_COMM_WORLD, &b_y_send_request);
          MPI_Isend(f_y_send_vec, lat_xzt12, MPI_DOUBLE, move_f, 3,
                    MPI_COMM_WORLD, &f_y_send_request);
        }
        // send z
        wilson_dslash_z_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_z_send_vec, f_z_send_vec);
        if (grid_z != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_z, grid_z);
          move_forward(move_f, grid_index_z, grid_z);
          move_b = node_rank + move_b * grid_t;
          move_f = node_rank + move_f * grid_t;
          MPI_Irecv(b_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_b, 5,
                    MPI_COMM_WORLD, &b_z_recv_request);
          MPI_Irecv(f_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_f, 4,
                    MPI_COMM_WORLD, &f_z_recv_request);
          MPI_Isend(b_z_send_vec, lat_xyt12, MPI_DOUBLE, move_b, 4,
                    MPI_COMM_WORLD, &b_z_send_request);
          MPI_Isend(f_z_send_vec, lat_xyt12, MPI_DOUBLE, move_f, 5,
                    MPI_COMM_WORLD, &f_z_send_request);
        }
        // send t
        wilson_dslash_t_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_t_send_vec, f_t_send_vec);
        if (grid_t != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_t, grid_t);
          move_forward(move_f, grid_index_t, grid_t);
          move_b = node_rank + move_b;
          move_f = node_rank + move_f;
          MPI_Irecv(b_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_b, 7,
                    MPI_COMM_WORLD, &b_t_recv_request);
          MPI_Irecv(f_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_f, 6,
                    MPI_COMM_WORLD, &f_t_recv_request);
          MPI_Isend(b_t_send_vec, lat_xyz12, MPI_DOUBLE, move_b, 6,
                    MPI_COMM_WORLD, &b_t_send_request);
          MPI_Isend(f_t_send_vec, lat_xyz12, MPI_DOUBLE, move_f, 7,
                    MPI_COMM_WORLD, &f_t_send_request);
        }
        // recv x
        if (grid_x != 1) {
          MPI_Wait(&b_x_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_x_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_x_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_x_recv_vec,
              f_x_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_x_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_x_send_vec,
              b_x_send_vec);
        }
        // recv y
        if (grid_y != 1) {
          MPI_Wait(&b_y_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_y_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_y_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_y_recv_vec,
              f_y_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_y_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_y_send_vec,
              b_y_send_vec);
        }
        // recv z
        if (grid_z != 1) {
          MPI_Wait(&b_z_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_z_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_z_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_z_recv_vec,
              f_z_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_z_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_z_send_vec,
              b_z_send_vec);
        }
        // recv t
        if (grid_t != 1) {
          MPI_Wait(&b_t_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_t_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_t_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_t_recv_vec,
              f_t_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_t_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_t_send_vec,
              b_t_send_vec);
        }
        MPI_Barrier(MPI_COMM_WORLD);
      }
      // kappa
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          cg_out[i] = cg_out[i] - cg_in[i] * Kappa;
        }
      }    
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          local_result = r_tilde[i].conj() * v[i];
        }
        MPI_Allreduce(&local_result, &tmp, 2, MPI_DOUBLE, MPI_SUM,
                      MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
      }
      alpha = rho / tmp;
#ifdef DEBUG
      std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
                << "##alpha:" << alpha.real << std::endl;
#endif
      for (int i = 0; i < lat_xyzt12; i++) {
        s[i] = r[i] - v[i] * alpha;
      }
      // t = A * s;
      cg_in = s;
      cg_out = t;
      // mpi_wilson_dslash
      {
        // clean
        wilson_dslash_clear_dest<<<gridDim, blockDim>>>(cg_out, lat_x, lat_y,
                                                        lat_z);
        // send x
        wilson_dslash_x_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_x_send_vec, f_x_send_vec);
        if (grid_x != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_x, grid_x);
          move_forward(move_f, grid_index_x, grid_x);
          move_b = node_rank + move_b * grid_y * grid_z * grid_t;
          move_f = node_rank + move_f * grid_y * grid_z * grid_t;
          MPI_Irecv(b_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_b, 1,
                    MPI_COMM_WORLD, &b_x_recv_request);
          MPI_Irecv(f_x_recv_vec, lat_yzt12, MPI_DOUBLE, move_f, 0,
                    MPI_COMM_WORLD, &f_x_recv_request);
          MPI_Isend(b_x_send_vec, lat_yzt12, MPI_DOUBLE, move_b, 0,
                    MPI_COMM_WORLD, &b_x_send_request);
          MPI_Isend(f_x_send_vec, lat_yzt12, MPI_DOUBLE, move_f, 1,
                    MPI_COMM_WORLD, &f_x_send_request);
        }
        // send y
        wilson_dslash_y_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_y_send_vec, f_y_send_vec);
        if (grid_y != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_y, grid_y);
          move_forward(move_f, grid_index_y, grid_y);
          move_b = node_rank + move_b * grid_z * grid_t;
          move_f = node_rank + move_f * grid_z * grid_t;
          MPI_Irecv(b_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_b, 3,
                    MPI_COMM_WORLD, &b_y_recv_request);
          MPI_Irecv(f_y_recv_vec, lat_xzt12, MPI_DOUBLE, move_f, 2,
                    MPI_COMM_WORLD, &f_y_recv_request);
          MPI_Isend(b_y_send_vec, lat_xzt12, MPI_DOUBLE, move_b, 2,
                    MPI_COMM_WORLD, &b_y_send_request);
          MPI_Isend(f_y_send_vec, lat_xzt12, MPI_DOUBLE, move_f, 3,
                    MPI_COMM_WORLD, &f_y_send_request);
        }
        // send z
        wilson_dslash_z_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_z_send_vec, f_z_send_vec);
        if (grid_z != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_z, grid_z);
          move_forward(move_f, grid_index_z, grid_z);
          move_b = node_rank + move_b * grid_t;
          move_f = node_rank + move_f * grid_t;
          MPI_Irecv(b_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_b, 5,
                    MPI_COMM_WORLD, &b_z_recv_request);
          MPI_Irecv(f_z_recv_vec, lat_xyt12, MPI_DOUBLE, move_f, 4,
                    MPI_COMM_WORLD, &f_z_recv_request);
          MPI_Isend(b_z_send_vec, lat_xyt12, MPI_DOUBLE, move_b, 4,
                    MPI_COMM_WORLD, &b_z_send_request);
          MPI_Isend(f_z_send_vec, lat_xyt12, MPI_DOUBLE, move_f, 5,
                    MPI_COMM_WORLD, &f_z_send_request);
        }
        // send t
        wilson_dslash_t_send<<<gridDim, blockDim>>>(gauge, cg_in, cg_out, lat_x,
                                                    lat_y, lat_z, lat_t, parity,
                                                    b_t_send_vec, f_t_send_vec);
        if (grid_t != 1) {
          checkCudaErrors(hipDeviceSynchronize());
          move_backward(move_b, grid_index_t, grid_t);
          move_forward(move_f, grid_index_t, grid_t);
          move_b = node_rank + move_b;
          move_f = node_rank + move_f;
          MPI_Irecv(b_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_b, 7,
                    MPI_COMM_WORLD, &b_t_recv_request);
          MPI_Irecv(f_t_recv_vec, lat_xyz12, MPI_DOUBLE, move_f, 6,
                    MPI_COMM_WORLD, &f_t_recv_request);
          MPI_Isend(b_t_send_vec, lat_xyz12, MPI_DOUBLE, move_b, 6,
                    MPI_COMM_WORLD, &b_t_send_request);
          MPI_Isend(f_t_send_vec, lat_xyz12, MPI_DOUBLE, move_f, 7,
                    MPI_COMM_WORLD, &f_t_send_request);
        }
        // recv x
        if (grid_x != 1) {
          MPI_Wait(&b_x_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_x_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_x_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_x_recv_vec,
              f_x_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_x_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_x_send_vec,
              b_x_send_vec);
        }
        // recv y
        if (grid_y != 1) {
          MPI_Wait(&b_y_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_y_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_y_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_y_recv_vec,
              f_y_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_y_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_y_send_vec,
              b_y_send_vec);
        }
        // recv z
        if (grid_z != 1) {
          MPI_Wait(&b_z_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_z_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_z_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_z_recv_vec,
              f_z_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_z_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_z_send_vec,
              b_z_send_vec);
        }
        // recv t
        if (grid_t != 1) {
          MPI_Wait(&b_t_recv_request, MPI_STATUS_IGNORE);
          MPI_Wait(&f_t_recv_request, MPI_STATUS_IGNORE);
          wilson_dslash_t_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, b_t_recv_vec,
              f_t_recv_vec);
        } else {
          checkCudaErrors(hipDeviceSynchronize());
          wilson_dslash_t_recv<<<gridDim, blockDim>>>(
              gauge, cg_out, lat_x, lat_y, lat_z, lat_t, parity, f_t_send_vec,
              b_t_send_vec);
        }
        MPI_Barrier(MPI_COMM_WORLD);
      }
      // kappa
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          cg_out[i] = cg_out[i] - cg_in[i] * Kappa;
        }
      }
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          local_result = t[i].conj() * s[i];
        }
        MPI_Allreduce(&local_result, &tmp0, 2, MPI_DOUBLE, MPI_SUM,
                      MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
      }
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          local_result = t[i].conj() * t[i];
        }
        MPI_Allreduce(&local_result, &tmp1, 2, MPI_DOUBLE, MPI_SUM,
                      MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
      }
      omega = tmp0 / tmp1;
#ifdef DEBUG
      std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
                << "##omega:" << omega.real << std::endl;
#endif
      for (int i = 0; i < lat_xyzt12; i++) {
        x[i] = x[i] + p[i] * alpha + s[i] * omega;
      }
      for (int i = 0; i < lat_xyzt12; i++) {
        r[i] = s[i] - t[i] * omega;
      }
      {
        for (int i = 0; i < lat_xyzt12; i++) {
          local_result = r[i].conj() * r[i];
        }
        MPI_Allreduce(&local_result, &r_norm2, 2, MPI_DOUBLE, MPI_SUM,
                      MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
      }
      std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
                << "##Residual:" << r_norm2.real << std::endl;
      // break;
      if (r_norm2.real < TOL || loop == MAX_ITER - 1) {
        break;
      }
      rho_prev = rho;
    }
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    err = hipGetLastError();
    checkCudaErrors(err);
    printf("mpi wilson cg total time: (without malloc free memcpy) :%.9lf "
           "sec\n",
           double(duration) / 1e9);
    {
      // free
      checkCudaErrors(hipFree(b_x_send_vec));
      checkCudaErrors(hipFree(f_x_send_vec));
      checkCudaErrors(hipFree(b_y_send_vec));
      checkCudaErrors(hipFree(f_y_send_vec));
      checkCudaErrors(hipFree(b_z_send_vec));
      checkCudaErrors(hipFree(f_z_send_vec));
      checkCudaErrors(hipFree(b_t_send_vec));
      checkCudaErrors(hipFree(f_t_send_vec));
      checkCudaErrors(hipFree(b_x_recv_vec));
      checkCudaErrors(hipFree(f_x_recv_vec));
      checkCudaErrors(hipFree(b_y_recv_vec));
      checkCudaErrors(hipFree(f_y_recv_vec));
      checkCudaErrors(hipFree(b_z_recv_vec));
      checkCudaErrors(hipFree(f_z_recv_vec));
      checkCudaErrors(hipFree(b_t_recv_vec));
      checkCudaErrors(hipFree(f_t_recv_vec));
      checkCudaErrors(hipFree(x));
      checkCudaErrors(hipFree(b));
      checkCudaErrors(hipFree(r));
      checkCudaErrors(hipFree(r_tilde));
      checkCudaErrors(hipFree(p));
      checkCudaErrors(hipFree(v));
      checkCudaErrors(hipFree(s));
      checkCudaErrors(hipFree(t));
    }
  }
}
#endif