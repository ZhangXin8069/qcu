#include <iostream>
#pragma optimize(5)
#include "../../include/qcu.h"
#ifdef MPI_WILSON_CG
#define DEBUG_MPI_WILSON_CG
#define TEST_MPI_WILSON_CG 0
#define TEST_MPI_WILSON_CG_USE_WILSON_DSLASH 0

void mpiBistabCgQcu(void *gauge, QcuParam *param, QcuParam *grid) {
  // define for mpi_wilson_dslash
  int lat_1dim[DIM];
  int lat_3dim[DIM];
  int lat_4dim;
  give_dims(param, lat_1dim, lat_3dim, lat_4dim);
  int lat_3dim6[DIM];
  int lat_3dim12[DIM];
  for (int i = 0; i < DIM; i++) {
    lat_3dim6[i] = lat_3dim[i] * 6;
    lat_3dim12[i] = lat_3dim6[i] * 2;
  }
  hipError_t err;
  dim3 gridDim(lat_4dim / BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  int node_rank;
  int move[BF];
  int grid_1dim[DIM];
  int grid_index_1dim[DIM];
  give_grid(grid, node_rank, grid_1dim, grid_index_1dim);
  MPI_Request send_request[WARDS];
  MPI_Request recv_request[WARDS];
  void *send_vec[WARDS];
  void *recv_vec[WARDS];
  malloc_recv(lat_3dim6, send_vec, recv_vec);
  // define end
  // define for mpi_wilson_cg
  int lat_4dim12 = lat_4dim * 12;
  LatticeComplex *dslash_in, *dslash_out, *x, *b, *r, *r_tilde, *p, *v, *s, *t;
  hipMallocManaged(&x, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&b, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&r, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&r_tilde, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&p, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&v, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&s, lat_4dim12 * sizeof(LatticeComplex));
  hipMallocManaged(&t, lat_4dim12 * sizeof(LatticeComplex));
  LatticeComplex r_norm2(0.0, 0.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex one(1.0, 0.0);
  const int MAX_ITER(1e2); // 300++?
  const double TOL(1e-6);
  LatticeComplex rho_prev(1.0, 0.0);
  LatticeComplex rho(0.0, 0.0);
  LatticeComplex alpha(1.0, 0.0);
  LatticeComplex omega(1.0, 0.0);
  LatticeComplex beta(0.0, 0.0);
  LatticeComplex tmp(0.0, 0.0);
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex local_result(0.0, 0.0);
  // double Kappa = 0.125;
  double Kappa = 10;
  auto start = std::chrono::high_resolution_clock::now();
  give_rand(x, lat_4dim12); // rand x
  // give_value(x, zero, lat_4dim12 );    // zero x
  // give_rand(b, lat_4dim12 );           // rand b
  give_value(b, one, 1);                 // point b
  give_value(r, zero, lat_4dim12);       // zero r
  give_value(r_tilde, zero, lat_4dim12); // zero r_tilde
  give_value(p, zero, lat_4dim12);       // zero p
  give_value(v, zero, lat_4dim12);       // zero v
  give_value(s, zero, lat_4dim12);       // zero s
  give_value(t, zero, lat_4dim12);       // zero t
  dslash_in = x;
  dslash_out = r;
  // define end
  _dslash(gridDim, blockDim, gauge, fermion_in, fermion_out, parity, lat_1dim,
          lat_3dim12, node_rank, grid_1dim, grid_index_1dim, move, send_request,
          recv_request, send_vec, recv_vec, dslash_in, dslash_out, Kappa, zero,
          one);
  for (int i = 0; i < lat_4dim12; i++) {
    r[i] = b[i] - r[i];
    r_tilde[i] = r[i];
  }
  for (int loop = 0; loop < MAX_ITER; loop++) {
    cg_mpi_dot(local_result, lat_4dim12, r_tilde, r, rho, zero);
#ifdef DEBUG_MPI_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##rho:" << rho.real << std::endl;
#endif
    beta = (rho / rho_prev) * (alpha / omega);
#ifdef DEBUG_MPI_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##beta:" << beta.real << std::endl;
#endif
    for (int i = 0; i < lat_4dim12; i++) {
      p[i] = r[i] + (p[i] - v[i] * omega) * beta;
    }
    // v = A * p;
    dslash_in = p;
    dslash_out = v;
    _dslash(gridDim, blockDim, gauge, fermion_in, fermion_out, parity, lat_1dim,
            lat_3dim12, node_rank, grid_1dim, grid_index_1dim, move,
            send_request, recv_request, send_vec, recv_vec, dslash_in,
            dslash_out, Kappa, zero, one);
    cg_mpi_dot(local_result, lat_4dim12, r_tilde, v, tmp, zero);
    alpha = rho / tmp;
#ifdef DEBUG_MPI_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##alpha:" << alpha.real << std::endl;
#endif
    for (int i = 0; i < lat_4dim12; i++) {
      s[i] = r[i] - v[i] * alpha;
    }
    // t = A * s;
    dslash_in = s;
    dslash_out = t;
    _dslash(gridDim, blockDim, gauge, fermion_in, fermion_out, parity, lat_1dim,
            lat_3dim12, node_rank, grid_1dim, grid_index_1dim, move,
            send_request, recv_request, send_vec, recv_vec, dslash_in,
            dslash_out, Kappa, zero, one);
    cg_mpi_dot(local_result, lat_4dim12, t, s, tmp0, zero);
    cg_mpi_dot(local_result, lat_4dim12, t, t, tmp1, zero);
    omega = tmp0 / tmp1;
#ifdef DEBUG_MPI_WILSON_CG
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##omega:" << omega.real << std::endl;
#endif
    for (int i = 0; i < lat_4dim12; i++) {
      x[i] = x[i] + p[i] * alpha + s[i] * omega;
    }
    for (int i = 0; i < lat_4dim12; i++) {
      r[i] = s[i] - t[i] * omega;
    }
    cg_mpi_dot(local_result, lat_4dim12, r, r, r_norm2, zero);
    std::cout << "##RANK:" << node_rank << "##LOOP:" << loop
              << "##Residual:" << r_norm2.real << std::endl;
    // break;
    if (r_norm2.real < TOL || loop == MAX_ITER - 1) {
      break;
    }
    rho_prev = rho;
  }
  checkCudaErrors(hipDeviceSynchronize());
  auto end = std::chrono::high_resolution_clock::now();
  auto duration =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  err = hipGetLastError();
  checkCudaErrors(err);
  printf("mpi wilson cg total time: (without malloc free "
         "memcpy) :%.9lf "
         "sec\n",
         double(duration) / 1e9);
  // free
  free_recv(send_vec, recv_vec);
  hipFree(x);
  hipFree(b);
  hipFree(r);
  hipFree(r_tilde);
  hipFree(p);
  hipFree(v);
  hipFree(s);
  hipFree(t);
}

#endif