#include "hip/hip_runtime.h"
#pragma optimize(5)
#include "../../include/qcu.h"
__global__ void wilson_dslash_clear_dest(void *device_dest, int device_lat_x,
                                         const int device_lat_y,
                                         const int device_lat_z) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) +
       t * lat_z * lat_y * lat_x * 12 + z * lat_y * lat_x * 12 +
       y * lat_x * 12 + x * 12);
  give_value(origin_dest, zero, 12);
}
__global__ void
wilson_dslash_x_send(void *device_U, void *device_src, void *device_dest,
                     int device_lat_x, const int device_lat_y,
                     const int device_lat_z, const int device_lat_t,
                     const int device_parity, void *device_b_x_send_vec,
                     void *device_f_x_send_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  const int lat_xcc = lat_x * 9;
  const int lat_yxcc = lat_y * lat_xcc;
  const int lat_zyxcc = lat_z * lat_yxcc;
  const int lat_tzyxcc = lat_t * lat_zyxcc;
  const int lat_xsc = lat_x * 12;
  const int lat_yxsc = lat_y * lat_xsc;
  const int lat_zyxsc = lat_z * lat_yxsc;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  const int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) + t * lat_zyxcc +
       z * lat_yxcc + y * lat_xcc + x * 9);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_b_x_send_vec =
      ((static_cast<LatticeComplex *>(device_b_x_send_vec)) +
       (t * lat_z * lat_y + z * lat_y + y) * 6);
  LatticeComplex *origin_f_x_send_vec =
      ((static_cast<LatticeComplex *>(device_f_x_send_vec)) +
       (t * lat_z * lat_y + z * lat_y + y) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex src[12];
  LatticeComplex dest[12];
  LatticeComplex b_x_send_vec[6];
  LatticeComplex f_x_send_vec[6];
  give_value(dest, zero, 12);
  {
    // x-1
    move_backward_x(move, x, lat_x, eo, parity);
    if (move != lat_x - 1) {
      tmp_U = (origin_U + move * 9 + (1 - parity) * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * 12);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp1 * I;
          dest[c0 + 9] -= tmp0 * I;
        }
      }
    }
    if (x == 0 && move == 0) { // even-odd
      // send in x+1 way
      give_ptr(src, origin_src, 12);
      {
        // sigma src
        for (int c1 = 0; c1 < 3; c1++) {
          b_x_send_vec[c1] = src[c1] - src[c1 + 9] * I;
          b_x_send_vec[c1 + 3] = src[c1 + 3] - src[c1 + 6] * I;
        }
        give_ptr(origin_b_x_send_vec, b_x_send_vec, 6);
      }
    }
  }
  {
    // x+1
    move_forward_x(move, x, lat_x, eo, parity);
    if (move != 1 - lat_x) {
      tmp_U = (origin_U + parity * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * 12);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] - src[c1 + 9] * I) * U[c0 * 3 + c1];
            tmp1 += (src[c1 + 3] - src[c1 + 6] * I) * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp1 * I;
          dest[c0 + 9] += tmp0 * I;
        }
      }
    }
    if (x == lat_x - 1 && move == 0) { // even-odd
      // send in x-1 way
      tmp_U = (origin_U + (1 - parity) * lat_tzyxcc); // even-odd
      give_u(U, tmp_U);
      give_ptr(src, origin_src, 12);
      {
        // just tmp
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
          }
          f_x_send_vec[c0] = tmp0;
          f_x_send_vec[c0 + 3] = tmp1;
        }
        give_ptr(origin_f_x_send_vec, f_x_send_vec, 6);
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_x_recv(void *device_U, void *device_dest, int device_lat_x,
                     const int device_lat_y, const int device_lat_z,
                     const int device_lat_t, const int device_parity,
                     void *device_b_x_recv_vec, void *device_f_x_recv_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  const int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) +
       t * lat_z * lat_y * lat_x * 9 + z * lat_y * lat_x * 9 + y * lat_x * 9 +
       x * 9);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) +
       t * lat_z * lat_y * lat_x * 12 + z * lat_y * lat_x * 12 +
       y * lat_x * 12 + x * 12);
  LatticeComplex *origin_b_x_recv_vec =
      ((static_cast<LatticeComplex *>(device_b_x_recv_vec)) +
       (t * lat_z * lat_y + z * lat_y + y) * 6);
  LatticeComplex *origin_f_x_recv_vec =
      ((static_cast<LatticeComplex *>(device_f_x_recv_vec)) +
       (t * lat_z * lat_y + z * lat_y + y) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex dest[12];
  LatticeComplex b_x_recv_vec[6];
  LatticeComplex f_x_recv_vec[6];
  // needed
  give_value(dest, zero, 12);
  {
    // x-1
    move_backward_x(move, x, lat_x, eo, parity);
    if (move == lat_x - 1) {
      // recv in x-1 way
      give_ptr(b_x_recv_vec, origin_b_x_recv_vec, 6);
      for (int c0 = 0; c0 < 3; c0++) {
        dest[c0] += b_x_recv_vec[c0];
        dest[c0 + 3] += b_x_recv_vec[c0 + 3];
        dest[c0 + 6] -= b_x_recv_vec[c0 + 3] * I;
        dest[c0 + 9] -= b_x_recv_vec[c0] * I;
      }
    }
  }
  {
    // x+1
    move_forward_x(move, x, lat_x, eo, parity);
    if (move == 1 - lat_x) {
      // recv in x+1 way
      give_ptr(f_x_recv_vec, origin_f_x_recv_vec, 6);
      tmp_U = (origin_U + parity * lat_t * lat_z * lat_y * lat_x * 9);
      give_u(U, tmp_U);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += f_x_recv_vec[c1] * U[c0 * 3 + c1];
            tmp1 += f_x_recv_vec[c1 + 3] * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp1 * I;
          dest[c0 + 9] += tmp0 * I;
        }
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_y_send(void *device_U, void *device_src, void *device_dest,
                     int device_lat_x, const int device_lat_y,
                     const int device_lat_z, const int device_lat_t,
                     const int device_parity, void *device_b_y_send_vec,
                     void *device_f_y_send_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  const int lat_xcc = lat_x * 9;
  const int lat_yxcc = lat_y * lat_xcc;
  const int lat_zyxcc = lat_z * lat_yxcc;
  const int lat_tzyxcc = lat_t * lat_zyxcc;
  const int lat_xsc = lat_x * 12;
  const int lat_yxsc = lat_y * lat_xsc;
  const int lat_zyxsc = lat_z * lat_yxsc;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) + t * lat_zyxcc +
       z * lat_yxcc + y * lat_xcc + x * 9);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_b_y_send_vec =
      ((static_cast<LatticeComplex *>(device_b_y_send_vec)) +
       (t * lat_z * lat_x + z * lat_x + x) * 6);
  LatticeComplex *origin_f_y_send_vec =
      ((static_cast<LatticeComplex *>(device_f_y_send_vec)) +
       (t * lat_z * lat_x + z * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex src[12];
  LatticeComplex dest[12];
  LatticeComplex b_y_send_vec[6];
  LatticeComplex f_y_send_vec[6];
  give_value(dest, zero, 12);
  {
    // y-1
    move_backward(move, y, lat_y);
    if (move == -1) {
      tmp_U = (origin_U + move * lat_xcc + lat_tzyxcc * 2 +
               (1 - parity) * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_xsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] - src[c1 + 9]) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp1;
          dest[c0 + 9] -= tmp0;
        }
      }
    } else {
      // send in y+1 way
      give_ptr(src, origin_src, 12);
      {
        // sigma src
        for (int c1 = 0; c1 < 3; c1++) {
          b_y_send_vec[c1] = src[c1] + src[c1 + 9];
          b_y_send_vec[c1 + 3] = src[c1 + 3] - src[c1 + 6];
        }
        give_ptr(origin_b_y_send_vec, b_y_send_vec, 6);
      }
    }
  }
  {
    // y+1
    move_forward(move, y, lat_y);
    if (move == 1) {
      tmp_U = (origin_U + lat_tzyxcc * 2 + parity * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_xsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 9]) * U[c0 * 3 + c1];
            tmp1 += (src[c1 + 3] - src[c1 + 6]) * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp1;
          dest[c0 + 9] += tmp0;
        }
      }
    } else {
      // send in y-1 way
      tmp_U =
          (origin_U + +lat_tzyxcc * 2 + (1 - parity) * lat_tzyxcc); // even-odd
      give_u(U, tmp_U);
      give_ptr(src, origin_src, 12);
      {
        // just tmp
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] - src[c1 + 9]) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
          }
          f_y_send_vec[c0] = tmp0;
          f_y_send_vec[c0 + 3] = tmp1;
        }
        give_ptr(origin_f_y_send_vec, f_y_send_vec, 6);
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_y_recv(void *device_U, void *device_dest, int device_lat_x,
                     const int device_lat_y, const int device_lat_z,
                     const int device_lat_t, const int device_parity,
                     void *device_b_y_recv_vec, void *device_f_y_recv_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) +
       t * lat_z * lat_y * lat_x * 9 + z * lat_y * lat_x * 9 + y * lat_x * 9 +
       x * 9);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) +
       t * lat_z * lat_y * lat_x * 12 + z * lat_y * lat_x * 12 +
       y * lat_x * 12 + x * 12);
  LatticeComplex *origin_b_y_recv_vec =
      ((static_cast<LatticeComplex *>(device_b_y_recv_vec)) +
       (t * lat_z * lat_x + z * lat_x + x) * 6);
  LatticeComplex *origin_f_y_recv_vec =
      ((static_cast<LatticeComplex *>(device_f_y_recv_vec)) +
       (t * lat_z * lat_x + z * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex dest[12];
  LatticeComplex b_y_recv_vec[6];
  LatticeComplex f_y_recv_vec[6];
  // needed
  give_value(dest, zero, 12);
  {
    // y-1
    move_backward(move, y, lat_y);
    if (move != -1) {
      // recv in y-1 way
      give_ptr(b_y_recv_vec, origin_b_y_recv_vec, 6);
      for (int c0 = 0; c0 < 3; c0++) {
        dest[c0] += b_y_recv_vec[c0];
        dest[c0 + 3] += b_y_recv_vec[c0 + 3];
        dest[c0 + 6] += b_y_recv_vec[c0 + 3];
        dest[c0 + 9] -= b_y_recv_vec[c0];
      }
    }
  }
  {
    // y+1
    move_forward(move, y, lat_y);
    if (move != 1) {
      // recv in y+1 way
      give_ptr(f_y_recv_vec, origin_f_y_recv_vec, 6);
      tmp_U = (origin_U + (parity + 2) * lat_t * lat_z * lat_y * lat_x * 9);
      give_u(U, tmp_U);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += f_y_recv_vec[c1] * U[c0 * 3 + c1];
            tmp1 += f_y_recv_vec[c1 + 3] * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp1;
          dest[c0 + 9] += tmp0;
        }
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_z_send(void *device_U, void *device_src, void *device_dest,
                     int device_lat_x, const int device_lat_y,
                     const int device_lat_z, const int device_lat_t,
                     const int device_parity, void *device_b_z_send_vec,
                     void *device_f_z_send_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  const int lat_xcc = lat_x * 9;
  const int lat_yxcc = lat_y * lat_xcc;
  const int lat_zyxcc = lat_z * lat_yxcc;
  const int lat_tzyxcc = lat_t * lat_zyxcc;
  const int lat_xsc = lat_x * 12;
  const int lat_yxsc = lat_y * lat_xsc;
  const int lat_zyxsc = lat_z * lat_yxsc;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) + t * lat_zyxcc +
       z * lat_yxcc + y * lat_xcc + x * 9);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_b_z_send_vec =
      ((static_cast<LatticeComplex *>(device_b_z_send_vec)) +
       (t * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *origin_f_z_send_vec =
      ((static_cast<LatticeComplex *>(device_f_z_send_vec)) +
       (t * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex src[12];
  LatticeComplex dest[12];
  LatticeComplex b_z_send_vec[6];
  LatticeComplex f_z_send_vec[6];
  give_value(dest, zero, 12);
  {
    // z-1
    move_backward(move, z, lat_z);
    if (move == -1) {
      tmp_U = (origin_U + move * lat_yxcc + lat_tzyxcc * 4 +
               (1 - parity) * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_yxsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] - src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp0 * I;
          dest[c0 + 9] += tmp1 * I;
        }
      }
    } else {
      // send in z+1 way
      give_ptr(src, origin_src, 12);
      {
        // sigma src
        for (int c1 = 0; c1 < 3; c1++) {
          b_z_send_vec[c1] = src[c1] - src[c1 + 6] * I;
          b_z_send_vec[c1 + 3] = src[c1 + 3] + src[c1 + 9] * I;
        }
        give_ptr(origin_b_z_send_vec, b_z_send_vec, 6);
      }
    }
  }
  {
    // z+1
    move_forward(move, z, lat_z);
    if (move == 1) {
      tmp_U = (origin_U + lat_tzyxcc * 4 + parity * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_yxsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] - src[c1 + 6] * I) * U[c0 * 3 + c1];
            tmp1 += (src[c1 + 3] + src[c1 + 9] * I) * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp0 * I;
          dest[c0 + 9] -= tmp1 * I;
        }
      }
    } else {
      // send in z-1 way
      tmp_U =
          (origin_U + 4 * lat_tzyxcc + (1 - parity) * lat_tzyxcc); // even-odd
      give_u(U, tmp_U);
      give_ptr(src, origin_src, 12);
      {
        // just tmp
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 6] * I) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] - src[c1 + 9] * I) * U[c1 * 3 + c0].conj();
          }
          f_z_send_vec[c0] = tmp0;
          f_z_send_vec[c0 + 3] = tmp1;
        }
        give_ptr(origin_f_z_send_vec, f_z_send_vec, 6);
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_z_recv(void *device_U, void *device_dest, int device_lat_x,
                     const int device_lat_y, const int device_lat_z,
                     const int device_lat_t, const int device_parity,
                     void *device_b_z_recv_vec, void *device_f_z_recv_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) +
       t * lat_z * lat_y * lat_x * 9 + z * lat_y * lat_x * 9 + y * lat_x * 9 +
       x * 9);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) +
       t * lat_z * lat_y * lat_x * 12 + z * lat_y * lat_x * 12 +
       y * lat_x * 12 + x * 12);
  LatticeComplex *origin_b_z_recv_vec =
      ((static_cast<LatticeComplex *>(device_b_z_recv_vec)) +
       (t * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *origin_f_z_recv_vec =
      ((static_cast<LatticeComplex *>(device_f_z_recv_vec)) +
       (t * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex dest[12];
  LatticeComplex b_z_recv_vec[6];
  LatticeComplex f_z_recv_vec[6];
  // needed
  give_value(dest, zero, 12);
  {
    // z-1
    move_backward(move, z, lat_z);
    if (move != -1) {
      // recv in z-1 way
      give_ptr(b_z_recv_vec, origin_b_z_recv_vec, 6);
      for (int c0 = 0; c0 < 3; c0++) {
        dest[c0] += b_z_recv_vec[c0];
        dest[c0 + 3] += b_z_recv_vec[c0 + 3];
        dest[c0 + 6] -= b_z_recv_vec[c0] * I;
        dest[c0 + 9] += b_z_recv_vec[c0 + 3] * I;
      }
    }
  }
  {
    // z+1
    move_forward(move, z, lat_z);
    if (move != 1) {
      // recv in z+1 way
      give_ptr(f_z_recv_vec, origin_f_z_recv_vec, 6);
      tmp_U = (origin_U + (parity + 4) * lat_t * lat_z * lat_y * lat_x * 9);
      give_u(U, tmp_U);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += f_z_recv_vec[c1] * U[c0 * 3 + c1];
            tmp1 += f_z_recv_vec[c1 + 3] * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp0 * I;
          dest[c0 + 9] -= tmp1 * I;
        }
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_t_send(void *device_U, void *device_src, void *device_dest,
                     int device_lat_x, const int device_lat_y,
                     const int device_lat_z, const int device_lat_t,
                     const int device_parity, void *device_b_t_send_vec,
                     void *device_f_t_send_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  const int lat_xcc = lat_x * 9;
  const int lat_yxcc = lat_y * lat_xcc;
  const int lat_zyxcc = lat_z * lat_yxcc;
  const int lat_tzyxcc = lat_t * lat_zyxcc;
  const int lat_xsc = lat_x * 12;
  const int lat_yxsc = lat_y * lat_xsc;
  const int lat_zyxsc = lat_z * lat_yxsc;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) + t * lat_zyxcc +
       z * lat_yxcc + y * lat_xcc + x * 9);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + t * lat_zyxsc +
       z * lat_yxsc + y * lat_xsc + x * 12);
  LatticeComplex *origin_b_t_send_vec =
      ((static_cast<LatticeComplex *>(device_b_t_send_vec)) +
       (z * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *origin_f_t_send_vec =
      ((static_cast<LatticeComplex *>(device_f_t_send_vec)) +
       (z * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex src[12];
  LatticeComplex dest[12];
  LatticeComplex b_t_send_vec[6];
  LatticeComplex f_t_send_vec[6];
  give_value(dest, zero, 12);
  {
    // t-1
    move_backward(move, t, lat_t);
    if (move == -1) {
      tmp_U = (origin_U + move * lat_zyxcc + lat_tzyxcc * 6 +
               (1 - parity) * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_zyxsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 9]) * U[c1 * 3 + c0].conj();
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] += tmp0;
          dest[c0 + 9] += tmp1;
        }
      }
    } else {
      // send in t+1 way
      give_ptr(src, origin_src, 12);
      {
        // sigma src
        for (int c1 = 0; c1 < 3; c1++) {
          b_t_send_vec[c1] = src[c1] - src[c1 + 6];
          b_t_send_vec[c1 + 3] = src[c1 + 3] - src[c1 + 9];
        }
        give_ptr(origin_b_t_send_vec, b_t_send_vec, 6);
      }
    }
  }
  {
    // t+1
    move_forward(move, t, lat_t);
    if (move == 1) {
      tmp_U = (origin_U + lat_tzyxcc * 6 + parity * lat_tzyxcc);
      give_u(U, tmp_U);
      tmp_src = (origin_src + move * lat_zyxsc);
      give_ptr(src, tmp_src, 12);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] - src[c1 + 6]) * U[c0 * 3 + c1];
            tmp1 += (src[c1 + 3] - src[c1 + 9]) * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp0;
          dest[c0 + 9] -= tmp1;
        }
      }
    } else {
      // send in t-1 way
      tmp_U =
          (origin_U + lat_tzyxcc * 6 + (1 - parity) * lat_tzyxcc); // even-odd
      give_u(U, tmp_U);
      give_ptr(src, origin_src, 12);
      {
        // just tmp
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += (src[c1] + src[c1 + 6]) * U[c1 * 3 + c0].conj();
            tmp1 += (src[c1 + 3] + src[c1 + 9]) * U[c1 * 3 + c0].conj();
          }
          f_t_send_vec[c0] = tmp0;
          f_t_send_vec[c0 + 3] = tmp1;
        }
        give_ptr(origin_f_t_send_vec, f_t_send_vec, 6);
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
__global__ void
wilson_dslash_t_recv(void *device_U, void *device_dest, int device_lat_x,
                     const int device_lat_y, const int device_lat_z,
                     const int device_lat_t, const int device_parity,
                     void *device_b_t_recv_vec, void *device_f_t_recv_vec) {
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  const int lat_x = device_lat_x;
  const int lat_y = device_lat_y;
  const int lat_z = device_lat_z;
  const int lat_t = device_lat_t;
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U =
      ((static_cast<LatticeComplex *>(device_U)) +
       t * lat_z * lat_y * lat_x * 9 + z * lat_y * lat_x * 9 + y * lat_x * 9 +
       x * 9);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) +
       t * lat_z * lat_y * lat_x * 12 + z * lat_y * lat_x * 12 +
       y * lat_x * 12 + x * 12);
  LatticeComplex *origin_b_t_recv_vec =
      ((static_cast<LatticeComplex *>(device_b_t_recv_vec)) +
       (z * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *origin_f_t_recv_vec =
      ((static_cast<LatticeComplex *>(device_f_t_recv_vec)) +
       (z * lat_y * lat_x + y * lat_x + x) * 6);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[9];
  LatticeComplex dest[12];
  LatticeComplex b_t_recv_vec[6];
  LatticeComplex f_t_recv_vec[6];
  // needed
  give_value(dest, zero, 12);
  {
    // t-1
    move_backward(move, t, lat_t);
    if (move != -1) {
      // recv in t-1 way
      give_ptr(b_t_recv_vec, origin_b_t_recv_vec, 6);
      for (int c0 = 0; c0 < 3; c0++) {
        dest[c0] += b_t_recv_vec[c0];
        dest[c0 + 3] += b_t_recv_vec[c0 + 3];
        dest[c0 + 6] += b_t_recv_vec[c0];
        dest[c0 + 9] += b_t_recv_vec[c0 + 3];
      }
    }
  }
  {
    // t+1
    move_forward(move, t, lat_t);
    if (move != 1) {
      // recv in t+1 way
      give_ptr(f_t_recv_vec, origin_f_t_recv_vec, 6);
      tmp_U = (origin_U + (parity + 6) * lat_t * lat_z * lat_y * lat_x * 9);
      give_u(U, tmp_U);
      {
        for (int c0 = 0; c0 < 3; c0++) {
          tmp0 = zero;
          tmp1 = zero;
          for (int c1 = 0; c1 < 3; c1++) {
            tmp0 += f_t_recv_vec[c1] * U[c0 * 3 + c1];
            tmp1 += f_t_recv_vec[c1 + 3] * U[c0 * 3 + c1];
          }
          dest[c0] += tmp0;
          dest[c0 + 3] += tmp1;
          dest[c0 + 6] -= tmp0;
          dest[c0 + 9] -= tmp1;
        }
      }
    }
  }
  // just add
  add_ptr(origin_dest, dest, 12);
}
#ifdef MPI_WILSON_DSLASH
void mpiDslashQcu(void *fermion_out, void *fermion_in, void *gauge,
                  QcuParam *param, int parity, QcuParam *grid) {
  int lat_1dim[DIM];
  int lat_3dim6[DIM];
  int lat_3dim12[DIM];
  int lat_4dim12;
  give_dims(param, lat_1dim, lat_3dim6, lat_3dim12, lat_4dim12);
  hipError_t err;
  dim3 gridDim(lat_1dim[X] * lat_1dim[Y] * lat_1dim[Z] * lat_1dim[T] /
               BLOCK_SIZE);
  dim3 blockDim(BLOCK_SIZE);
  {
    // mpi wilson dslash
    int node_rank;
    int move[BF];
    int grid_1dim[DIM];
    int grid_index_1dim[DIM];
    give_grid(grid, node_rank, grid_1dim, grid_index_1dim);
    MPI_Request send_request[WARDS];
    MPI_Request recv_request[WARDS];
    void *send_vec[WARDS];
    void *recv_vec[WARDS];
    checkCudaErrors(hipMallocManaged(&send_vec[B_X], lat_3dim6[YZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[F_X], lat_3dim6[YZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[B_Y], lat_3dim6[XZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[F_Y], lat_3dim6[XZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[B_Z], lat_3dim6[XYT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[F_Z], lat_3dim6[XYT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[B_T], lat_3dim6[XYZ] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&send_vec[F_T], lat_3dim6[XYZ] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[B_X], lat_3dim6[YZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[F_X], lat_3dim6[YZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[B_Y], lat_3dim6[XZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[F_Y], lat_3dim6[XZT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[B_Z], lat_3dim6[XYT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[F_Z], lat_3dim6[XYT] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[B_T], lat_3dim6[XYZ] * sizeof(LatticeComplex)));
    checkCudaErrors(hipMallocManaged(&recv_vec[F_T], lat_3dim6[XYZ] * sizeof(LatticeComplex)));
    // above define for mpi_wilson_dslash
    auto start = std::chrono::high_resolution_clock::now();
    // clean
    wilson_dslash_clear_dest<<<gridDim, blockDim>>>(fermion_out, lat_1dim[X],
                                                    lat_1dim[Y], lat_1dim[Z]);
    // send x
    wilson_dslash_x_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
        lat_1dim[T], parity, send_vec[B_X], send_vec[F_X]);
    if (grid_1dim[X] != 1) {
      checkCudaErrors(hipDeviceSynchronize());
      move_backward(move[B], grid_index_1dim[X], grid_1dim[X]);
      move_forward(move[F], grid_index_1dim[X], grid_1dim[X]);
      move[B] =
          node_rank + move[B] * grid_1dim[Y] * grid_1dim[Z] * grid_1dim[T];
      move[F] =
          node_rank + move[F] * grid_1dim[Y] * grid_1dim[Z] * grid_1dim[T];
      MPI_Irecv(recv_vec[B_X], lat_3dim12[YZT], MPI_DOUBLE, move[B], F_X,
                MPI_COMM_WORLD, &recv_request[B_X]);
      MPI_Irecv(recv_vec[F_X], lat_3dim12[YZT], MPI_DOUBLE, move[F], B_X,
                MPI_COMM_WORLD, &recv_request[F_X]);
      MPI_Isend(send_vec[B_X], lat_3dim12[YZT], MPI_DOUBLE, move[B], B_X,
                MPI_COMM_WORLD, &send_request[B_X]);
      MPI_Isend(send_vec[F_X], lat_3dim12[YZT], MPI_DOUBLE, move[F], F_X,
                MPI_COMM_WORLD, &send_request[F_T]);
    }
    // send y
    wilson_dslash_y_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
        lat_1dim[T], parity, send_vec[B_Y], send_vec[F_Y]);
    if (grid_1dim[Y] != 1) {
      checkCudaErrors(hipDeviceSynchronize());
      move_backward(move[B], grid_index_1dim[Y], grid_1dim[Y]);
      move_forward(move[F], grid_index_1dim[Y], grid_1dim[Y]);
      move[B] = node_rank + move[B] * grid_1dim[Z] * grid_1dim[T];
      move[F] = node_rank + move[F] * grid_1dim[Z] * grid_1dim[T];
      MPI_Irecv(recv_vec[B_Y], lat_3dim12[XZT], MPI_DOUBLE, move[B], F_Y,
                MPI_COMM_WORLD, &recv_request[B_Y]);
      MPI_Irecv(recv_vec[F_Y], lat_3dim12[XZT], MPI_DOUBLE, move[F], B_Y,
                MPI_COMM_WORLD, &recv_request[F_Y]);
      MPI_Isend(send_vec[B_Y], lat_3dim12[XZT], MPI_DOUBLE, move[B], B_Y,
                MPI_COMM_WORLD, &send_request[B_Y]);
      MPI_Isend(send_vec[F_Y], lat_3dim12[XZT], MPI_DOUBLE, move[F], F_Y,
                MPI_COMM_WORLD, &send_request[F_Y]);
    }
    // send z
    wilson_dslash_z_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
        lat_1dim[T], parity, send_vec[B_Z], send_vec[F_Z]);
    if (grid_1dim[Z] != 1) {
      checkCudaErrors(hipDeviceSynchronize());
      move_backward(move[B], grid_index_1dim[Z], grid_1dim[Z]);
      move_forward(move[F], grid_index_1dim[Z], grid_1dim[Z]);
      move[B] = node_rank + move[B] * grid_1dim[T];
      move[F] = node_rank + move[F] * grid_1dim[T];
      MPI_Irecv(recv_vec[B_Z], lat_3dim12[XYT], MPI_DOUBLE, move[B], F_Z,
                MPI_COMM_WORLD, &recv_request[B_Z]);
      MPI_Irecv(recv_vec[F_Z], lat_3dim12[XYT], MPI_DOUBLE, move[F], B_Z,
                MPI_COMM_WORLD, &recv_request[F_Z]);
      MPI_Isend(send_vec[B_Z], lat_3dim12[XYT], MPI_DOUBLE, move[B], B_Z,
                MPI_COMM_WORLD, &send_request[B_Z]);
      MPI_Isend(send_vec[F_Z], lat_3dim12[XYT], MPI_DOUBLE, move[F], F_Z,
                MPI_COMM_WORLD, &send_request[F_Z]);
    }
    // send t
    wilson_dslash_t_send<<<gridDim, blockDim>>>(
        gauge, fermion_in, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
        lat_1dim[T], parity, send_vec[B_T], send_vec[F_T]);
    if (grid_1dim[T] != 1) {
      checkCudaErrors(hipDeviceSynchronize());
      move_backward(move[B], grid_index_1dim[T], grid_1dim[T]);
      move_forward(move[F], grid_index_1dim[T], grid_1dim[T]);
      move[B] = node_rank + move[B];
      move[F] = node_rank + move[F];
      MPI_Irecv(recv_vec[B_T], lat_3dim12[XYZ], MPI_DOUBLE, move[B], F_T,
                MPI_COMM_WORLD, &recv_request[B_T]);
      MPI_Irecv(recv_vec[F_T], lat_3dim12[XYZ], MPI_DOUBLE, move[F], B_T,
                MPI_COMM_WORLD, &recv_request[F_T]);
      MPI_Isend(send_vec[B_T], lat_3dim12[XYZ], MPI_DOUBLE, move[B], B_T,
                MPI_COMM_WORLD, &send_request[B_T]);
      MPI_Isend(send_vec[F_T], lat_3dim12[XYZ], MPI_DOUBLE, move[F], F_T,
                MPI_COMM_WORLD, &send_request[F_T]);
    }
    // recv x
    if (grid_1dim[X] != 1) {
      MPI_Wait(&recv_request[B_X], MPI_STATUS_IGNORE);
      MPI_Wait(&recv_request[F_X], MPI_STATUS_IGNORE);
      wilson_dslash_x_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, recv_vec[B_X], recv_vec[F_X]);
    } else {
      checkCudaErrors(hipDeviceSynchronize());
      wilson_dslash_x_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, send_vec[F_X], send_vec[B_X]);
    }
    // recv y
    if (grid_1dim[Y] != 1) {
      MPI_Wait(&recv_request[B_Y], MPI_STATUS_IGNORE);
      MPI_Wait(&recv_request[F_Y], MPI_STATUS_IGNORE);
      wilson_dslash_y_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, recv_vec[B_Y], recv_vec[F_Y]);
    } else {
      checkCudaErrors(hipDeviceSynchronize());
      wilson_dslash_y_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, send_vec[F_Y], send_vec[B_Y]);
    }
    // recv z
    if (grid_1dim[Z] != 1) {
      MPI_Wait(&recv_request[B_Z], MPI_STATUS_IGNORE);
      MPI_Wait(&recv_request[F_Z], MPI_STATUS_IGNORE);
      wilson_dslash_z_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, recv_vec[B_Z], recv_vec[F_Z]);
    } else {
      checkCudaErrors(hipDeviceSynchronize());
      wilson_dslash_z_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, send_vec[F_Z], send_vec[B_Z]);
    }
    // recv t
    if (grid_1dim[T] != 1) {
      MPI_Wait(&recv_request[B_T], MPI_STATUS_IGNORE);
      MPI_Wait(&recv_request[F_T], MPI_STATUS_IGNORE);
      wilson_dslash_t_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, recv_vec[B_T], recv_vec[F_T]);
    } else {
      checkCudaErrors(hipDeviceSynchronize());
      wilson_dslash_t_recv<<<gridDim, blockDim>>>(
          gauge, fermion_out, lat_1dim[X], lat_1dim[Y], lat_1dim[Z],
          lat_1dim[T], parity, send_vec[F_T], send_vec[B_T]);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    checkCudaErrors(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    auto duration =
        std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
            .count();
    err = hipGetLastError();
    checkCudaErrors(err);
    printf("mpi wilson dslash total time: (without malloc free memcpy) :%.9lf "
           "sec\n",
           double(duration) / 1e9);
    // free 
    {
      checkCudaErrors(hipFree(send_vec[B_X]));
      checkCudaErrors(hipFree(send_vec[F_X]));
      checkCudaErrors(hipFree(send_vec[B_Y]));
      checkCudaErrors(hipFree(send_vec[F_Y]));
      checkCudaErrors(hipFree(send_vec[B_Z]));
      checkCudaErrors(hipFree(send_vec[F_Z]));
      checkCudaErrors(hipFree(send_vec[B_T]));
      checkCudaErrors(hipFree(send_vec[F_T]));
      checkCudaErrors(hipFree(recv_vec[B_X]));
      checkCudaErrors(hipFree(recv_vec[F_X]));
      checkCudaErrors(hipFree(recv_vec[B_Y]));
      checkCudaErrors(hipFree(recv_vec[F_Y]));
      checkCudaErrors(hipFree(recv_vec[B_Z]));
      checkCudaErrors(hipFree(recv_vec[F_Z]));
      checkCudaErrors(hipFree(recv_vec[B_T]));
      checkCudaErrors(hipFree(recv_vec[F_T]));
    }
  }
}
#endif