#include "hip/hip_runtime.h"
#include "../include/qcu.h"
#include "define.h"
#include "wilson_dslash.h"
#ifdef WILSON_DSLASH
#define __X__
#define __Y__
#define __Z__
#define __T__
__global__ void wilson_dslash(void *device_U, void *device_src,
                              void *device_dest, void *device_lat_xyzt,
                              const int device_parity) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int parity = idx;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  const int lat_x = lat_xyzt[_X_];
  const int lat_y = lat_xyzt[_Y_];
  const int lat_z = lat_xyzt[_Z_];
  const int lat_t = lat_xyzt[_T_];
  const int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  const int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  const int z = parity / move;
  parity -= z * move;
  const int y = parity / lat_x;
  const int x = parity - y * lat_x;
  parity = device_parity;
  const int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U = ((static_cast<LatticeComplex *>(device_U)) + idx);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + idx);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + idx);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  // just wilson(Sum part)
#ifdef __X__
  { // x part
   {// x-1
    move_backward_x(move, x, lat_x, eo, parity);
  tmp_U = (origin_U + move + (_X_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_3C_] * I) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_] * I) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp1 * I;
    dest[c0 + _LAT_3C_] -= tmp0 * I;
  }
}
{
  // x+1
  move_forward_x(move, x, lat_x, eo, parity);
  tmp_U = (origin_U + (_X_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_3C_] * I) * U[c0 * _LAT_C_ + c1];
      tmp1 +=
          (src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_] * I) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp1 * I;
    dest[c0 + _LAT_3C_] += tmp0 * I;
  }
}
}
#endif
#ifdef __Y__
{ // y part
 {// y-1
  move_backward(move, y, lat_y);
tmp_U = (origin_U + move * lat_x + (_Y_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
give_u(U, tmp_U, lat_tzyx);
tmp_src = (origin_src + move * lat_x);
give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_3C_]) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_]) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp1;
    dest[c0 + _LAT_3C_] -= tmp0;
  }
}
{
  // y+1
  move_forward(move, y, lat_y);
  tmp_U = (origin_U + (_Y_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move * lat_x);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_3C_]) * U[c0 * _LAT_C_ + c1];
      tmp1 += (src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_]) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp1;
    dest[c0 + _LAT_3C_] += tmp0;
  }
}
}
#endif
#ifdef __Z__
{ // z part
 {// z-1
  move_backward(move, z, lat_z);
tmp_U = (origin_U + move * lat_y * lat_x +
         (_Z_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
give_u(U, tmp_U, lat_tzyx);
tmp_src = (origin_src + move * lat_y * lat_x);
give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_2C_] * I) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_] * I) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp0 * I;
    dest[c0 + _LAT_3C_] += tmp1 * I;
  }
}
{
  // z+1
  move_forward(move, z, lat_z);
  tmp_U = (origin_U + (_Z_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move * lat_y * lat_x);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_; c0++) {
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_2C_] * I) * U[c0 * _LAT_C_ + c1];
      tmp1 +=
          (src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_] * I) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp0 * I;
    dest[c0 + _LAT_3C_] -= tmp1 * I;
  }
}
}
#endif
#ifdef __T__
{ // t part
  {
    // t-1
    move_backward(move, t, lat_t);
    tmp_U = (origin_U + move * lat_z * lat_y * lat_x +
             (_T_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    tmp_src = (origin_src + move * lat_z * lat_y * lat_x);
    give_src(src, tmp_src, lat_tzyx);
  }
  {
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        tmp0 += (src[c1] + src[c1 + _LAT_2C_]) * U[c1 * _LAT_C_ + c0].conj();
        tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_]) *
                U[c1 * _LAT_C_ + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + _LAT_1C_] += tmp1;
      dest[c0 + _LAT_2C_] += tmp0;
      dest[c0 + _LAT_3C_] += tmp1;
    }
  }
  {
    // t+1
    move_forward(move, t, lat_t);
    tmp_U = (origin_U + (_T_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    tmp_src = (origin_src + move * lat_z * lat_y * lat_x);
    give_src(src, tmp_src, lat_tzyx);
  }
  {
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        tmp0 += (src[c1] - src[c1 + _LAT_2C_]) * U[c0 * _LAT_C_ + c1];
        tmp1 +=
            (src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_]) * U[c0 * _LAT_C_ + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + _LAT_1C_] += tmp1;
      dest[c0 + _LAT_2C_] -= tmp0;
      dest[c0 + _LAT_3C_] -= tmp1;
    }
  }
}
#endif
give_dest(origin_dest, dest, lat_tzyx);
}
__global__ void wilson_dslash_inside(void *device_U, void *device_src,
                                     void *device_dest, void *device_lat_xyzt,
                                     int device_parity) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int parity = idx;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  int lat_y = lat_xyzt[_Y_];
  int lat_z = lat_xyzt[_Z_];
  int lat_t = lat_xyzt[_T_];
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *origin_U = ((static_cast<LatticeComplex *>(device_U)) + idx);
  LatticeComplex *origin_src =
      ((static_cast<LatticeComplex *>(device_src)) + idx);
  LatticeComplex *origin_dest =
      ((static_cast<LatticeComplex *>(device_dest)) + idx);
  LatticeComplex *tmp_U;
  LatticeComplex *tmp_src;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  // just wilson(Sum part)
#ifdef __X__
  { // x part
   {// x-1
    move_backward_x(move, x, lat_x, eo, parity);
  tmp_U = (origin_U + move + (_X_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move != lat_x - 1); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_3C_] * I) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_] * I) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp1 * I;
    dest[c0 + _LAT_3C_] -= tmp0 * I;
  }
}
{
  // x+1
  move_forward_x(move, x, lat_x, eo, parity);
  tmp_U = (origin_U + (_X_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move != 1 - lat_x); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_3C_] * I) * U[c0 * _LAT_C_ + c1];
      tmp1 +=
          (src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_] * I) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp1 * I;
    dest[c0 + _LAT_3C_] += tmp0 * I;
  }
}
}
#endif
#ifdef __Y__
{ // y part
 {// y-1
  move_backward(move, y, lat_y);
tmp_U = (origin_U + move * lat_x + (_Y_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
give_u(U, tmp_U, lat_tzyx);
tmp_src = (origin_src + move * lat_x);
give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move == -1); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_3C_]) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_]) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp1;
    dest[c0 + _LAT_3C_] -= tmp0;
  }
}
{
  // y+1
  move_forward(move, y, lat_y);
  tmp_U = (origin_U + (_Y_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move * lat_x);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move == 1); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_3C_]) * U[c0 * _LAT_C_ + c1];
      tmp1 += (src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_]) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp1;
    dest[c0 + _LAT_3C_] += tmp0;
  }
}
}
#endif
#ifdef __Z__
{ // z part
 {// z-1
  move_backward(move, z, lat_z);
tmp_U = (origin_U + move * lat_y * lat_x +
         (_Z_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
give_u(U, tmp_U, lat_tzyx);
tmp_src = (origin_src + move * lat_y * lat_x);
give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move == -1); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] + src[c1 + _LAT_2C_] * I) * U[c1 * _LAT_C_ + c0].conj();
      tmp1 += (src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_] * I) *
              U[c1 * _LAT_C_ + c0].conj();
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] -= tmp0 * I;
    dest[c0 + _LAT_3C_] += tmp1 * I;
  }
}
{
  // z+1
  move_forward(move, z, lat_z);
  tmp_U = (origin_U + (_Z_ * _EVENODD_ + parity) * lat_tzyx);
  give_u(U, tmp_U, lat_tzyx);
  tmp_src = (origin_src + move * lat_y * lat_x);
  give_src(src, tmp_src, lat_tzyx);
}
{
  for (int c0 = 0; c0 < _LAT_C_ * (move == 1); c0++) { // just inside
    tmp0 = zero;
    tmp1 = zero;
    for (int c1 = 0; c1 < _LAT_C_; c1++) {
      tmp0 += (src[c1] - src[c1 + _LAT_2C_] * I) * U[c0 * _LAT_C_ + c1];
      tmp1 +=
          (src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_] * I) * U[c0 * _LAT_C_ + c1];
    }
    dest[c0] += tmp0;
    dest[c0 + _LAT_1C_] += tmp1;
    dest[c0 + _LAT_2C_] += tmp0 * I;
    dest[c0 + _LAT_3C_] -= tmp1 * I;
  }
}
}
#endif
#ifdef __T__
{
  // t part
  {
    // t-1
    move_backward(move, t, lat_t);
    tmp_U = (origin_U + move * lat_z * lat_y * lat_x +
             (_T_ * _EVENODD_ + (1 - parity)) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    tmp_src = (origin_src + move * lat_z * lat_y * lat_x);
    give_src(src, tmp_src, lat_tzyx);
  }
  {
    for (int c0 = 0; c0 < _LAT_C_ * (move == -1); c0++) { // just inside
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        tmp0 += (src[c1] + src[c1 + _LAT_2C_]) * U[c1 * _LAT_C_ + c0].conj();
        tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_]) *
                U[c1 * _LAT_C_ + c0].conj();
      }
      dest[c0] += tmp0;
      dest[c0 + _LAT_1C_] += tmp1;
      dest[c0 + _LAT_2C_] += tmp0;
      dest[c0 + _LAT_3C_] += tmp1;
    }
  }
  {
    // t+1
    move_forward(move, t, lat_t);
    tmp_U = (origin_U + (_T_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    tmp_src = (origin_src + move * lat_z * lat_y * lat_x);
    give_src(src, tmp_src, lat_tzyx);
  }
  {
    for (int c0 = 0; c0 < _LAT_C_ * (move == 1); c0++) { // just inside
      tmp0 = zero;
      tmp1 = zero;
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        tmp0 += (src[c1] - src[c1 + _LAT_2C_]) * U[c0 * _LAT_C_ + c1];
        tmp1 +=
            (src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_]) * U[c0 * _LAT_C_ + c1];
      }
      dest[c0] += tmp0;
      dest[c0 + _LAT_1C_] += tmp1;
      dest[c0 + _LAT_2C_] -= tmp0;
      dest[c0 + _LAT_3C_] -= tmp1;
    }
  }
}
#endif
give_dest(origin_dest, dest, lat_tzyx);
}
__global__ void wilson_dslash_x_send(void *device_U, void *device_src,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_x_send_vec,
                                     void *device_f_x_send_vec) {
#ifdef __X__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  // int lat_x = lat_xyzt[_X_];
  int lat_x = 1; // so let x=0 first, then x = lat_x -1
  int lat_y = lat_xyzt[_Y_];
  int lat_z = lat_xyzt[_Z_];
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_x_send_vec[_LAT_HALF_SC_];
  LatticeComplex f_x_send_vec[_LAT_HALF_SC_];
  LatticeComplex *origin_U;
  LatticeComplex *origin_src;
  LatticeComplex *origin_b_x_send_vec;
  LatticeComplex *origin_f_x_send_vec;
  {
    lat_x = lat_xyzt[_X_]; // give lat_size back
    x = 0;                 // b_x
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_x_send_vec =
        ((static_cast<LatticeComplex *>(device_b_x_send_vec)) +
         (((t * lat_z + z) * lat_y + y)));
  }
  { // x-1
    move_backward_x(move, x, lat_x, eo, parity);
    // even-odd
    // send in x+1 way
    give_src(src, origin_src, lat_tzyx);
    { // sigma src
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        b_x_send_vec[c1] = src[c1] - src[c1 + _LAT_3C_] * I;
        b_x_send_vec[c1 + _LAT_1C_] =
            src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_] * I;
      }
      give_send_x(origin_b_x_send_vec, b_x_send_vec, lat_tzyx / lat_x,
                  (move == 0));
    }
  }
  {
    x = lat_x - 1; // f_x
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_x_send_vec =
        ((static_cast<LatticeComplex *>(device_f_x_send_vec)) +
         (((t * lat_z + z) * lat_y + y)));
  }
  { // x+1
    move_forward_x(move, x, lat_x, eo, parity);
    // even-odd
    // send in x-1 way
    tmp_U =
        (origin_U + (_X_ * _EVENODD_ + (1 - parity)) * lat_tzyx); // even-odd
    give_u(U, tmp_U, lat_tzyx);
    give_src(src, origin_src, lat_tzyx);
    { // just tmp
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 +=
              (src[c1] + src[c1 + _LAT_3C_] * I) * U[c1 * _LAT_C_ + c0].conj();
          tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_] * I) *
                  U[c1 * _LAT_C_ + c0].conj();
        }
        f_x_send_vec[c0] = tmp0;
        f_x_send_vec[c0 + _LAT_1C_] = tmp1;
      }
      give_send_x(origin_f_x_send_vec, f_x_send_vec, lat_tzyx / lat_x,
                  (move == 0));
    }
  }
#endif
}
__global__ void wilson_dslash_x_recv(void *device_U, void *device_dest,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_x_recv_vec,
                                     void *device_f_x_recv_vec) {
#ifdef __X__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  // int lat_x = lat_xyzt[_X_];
  int lat_x = 1; // so let x=0 first, then x = lat_x -1
  int lat_y = lat_xyzt[_Y_];
  int lat_z = lat_xyzt[_Z_];
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  int eo = (y + z + t) & 0x01; // (y+z+t)%2
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_x_recv_vec[_LAT_HALF_SC_];
  LatticeComplex f_x_recv_vec[_LAT_HALF_SC_]; // needed
  LatticeComplex *origin_U;
  LatticeComplex *origin_dest;
  LatticeComplex *origin_b_x_recv_vec;
  LatticeComplex *origin_f_x_recv_vec;
  {
    lat_x = lat_xyzt[_X_]; // give lat_size back
    x = 0;                 // b_x
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_x_recv_vec =
        ((static_cast<LatticeComplex *>(device_b_x_recv_vec)) +
         (((t * lat_z + z) * lat_y + y)));
  }
  { // x-1
    move_backward_x(move, x, lat_x, eo, parity);
    // recv in x-1 way
    give_recv(b_x_recv_vec, origin_b_x_recv_vec, lat_tzyx / lat_x);
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      dest[c0] += b_x_recv_vec[c0];
      dest[c0 + _LAT_1C_] += b_x_recv_vec[c0 + _LAT_1C_];
      dest[c0 + _LAT_2C_] -= b_x_recv_vec[c0 + _LAT_1C_] * I;
      dest[c0 + _LAT_3C_] -= b_x_recv_vec[c0] * I;
    }
  }                                                             // just add
  add_dest_x(origin_dest, dest, lat_tzyx, (move == lat_x - 1)); // even-odd
  for (int i = 0; i < _LAT_SC_; i++) {
    dest[i] = zero;
  }
  {
    x = lat_x - 1; // f_x
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_x_recv_vec =
        ((static_cast<LatticeComplex *>(device_f_x_recv_vec)) +
         (((t * lat_z + z) * lat_y + y)));
  }
  { // x+1
    move_forward_x(move, x, lat_x, eo, parity);
    // recv in x+1 way
    give_recv(f_x_recv_vec, origin_f_x_recv_vec, lat_tzyx / lat_x);
    tmp_U = (origin_U + (_X_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    {
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += f_x_recv_vec[c1] * U[c0 * _LAT_C_ + c1];
          tmp1 += f_x_recv_vec[c1 + _LAT_1C_] * U[c0 * _LAT_C_ + c1];
        }
        dest[c0] += tmp0;
        dest[c0 + _LAT_1C_] += tmp1;
        dest[c0 + _LAT_2C_] += tmp1 * I;
        dest[c0 + _LAT_3C_] += tmp0 * I;
      }
    }
  }                                                             // just add
  add_dest_x(origin_dest, dest, lat_tzyx, (move == 1 - lat_x)); // even-odd
#endif
}
__global__ void wilson_dslash_y_send(void *device_U, void *device_src,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_y_send_vec,
                                     void *device_f_y_send_vec) {
#ifdef __Y__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  // int lat_y = yyztsc[_y_];
  int lat_y = 1; // so let y=0 first, then y = lat_y -1
  int lat_z = lat_xyzt[_Z_];
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_y_send_vec[_LAT_HALF_SC_];
  LatticeComplex f_y_send_vec[_LAT_HALF_SC_];
  LatticeComplex *origin_U;
  LatticeComplex *origin_src;
  LatticeComplex *origin_b_y_send_vec;
  LatticeComplex *origin_f_y_send_vec;
  {
    lat_y = lat_xyzt[_Y_]; // give lat_size back
    y = 0;                 // b_y
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_y_send_vec =
        ((static_cast<LatticeComplex *>(device_b_y_send_vec)) +
         (((t * lat_z + z)) * lat_x + x));
  }
  { // y-1
    // move_backward(move, y, lat_y);
    // send in y+1 way
    give_src(src, origin_src, lat_tzyx);
    { // sigma src
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        b_y_send_vec[c1] = src[c1] + src[c1 + _LAT_3C_];
        b_y_send_vec[c1 + _LAT_1C_] = src[c1 + _LAT_1C_] - src[c1 + _LAT_2C_];
      }
      give_send(origin_b_y_send_vec, b_y_send_vec, lat_tzyx / lat_y);
    }
  }
  {
    y = lat_y - 1; // f_y
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_y_send_vec =
        ((static_cast<LatticeComplex *>(device_f_y_send_vec)) +
         (((t * lat_z + z)) * lat_x + x));
  }
  { // y+1
    // move_forward(move, y, lat_y);
    // send in y-1 way
    tmp_U =
        (origin_U + (_Y_ * _EVENODD_ + (1 - parity)) * lat_tzyx); // even-odd
    give_u(U, tmp_U, lat_tzyx);
    give_src(src, origin_src, lat_tzyx);
    { // just tmp
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += (src[c1] - src[c1 + _LAT_3C_]) * U[c1 * _LAT_C_ + c0].conj();
          tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_2C_]) *
                  U[c1 * _LAT_C_ + c0].conj();
        }
        f_y_send_vec[c0] = tmp0;
        f_y_send_vec[c0 + _LAT_1C_] = tmp1;
      }
      give_send(origin_f_y_send_vec, f_y_send_vec, lat_tzyx / lat_y);
    }
  }
#endif
}
__global__ void wilson_dslash_y_recv(void *device_U, void *device_dest,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_y_recv_vec,
                                     void *device_f_y_recv_vec) {
#ifdef __Y__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  // int lat_y = yyztsc[_y_];
  int lat_y = 1; // so let y=0 first, then y = lat_y -1
  int lat_z = lat_xyzt[_Z_];
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_y_recv_vec[_LAT_HALF_SC_];
  LatticeComplex f_y_recv_vec[_LAT_HALF_SC_]; // needed
  LatticeComplex *origin_U;
  LatticeComplex *origin_dest;
  LatticeComplex *origin_b_y_recv_vec;
  LatticeComplex *origin_f_y_recv_vec;
  {
    lat_y = lat_xyzt[_Y_]; // give lat_size back
    y = 0;                 // b_y
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_y_recv_vec =
        ((static_cast<LatticeComplex *>(device_b_y_recv_vec)) +
         (((t * lat_z + z)) * lat_x + x));
  }
  { // y-1
    move_backward(move, y, lat_y);
    // recv in y-1 way
    give_recv(b_y_recv_vec, origin_b_y_recv_vec, lat_tzyx / lat_y);
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      dest[c0] += b_y_recv_vec[c0];
      dest[c0 + _LAT_1C_] += b_y_recv_vec[c0 + _LAT_1C_];
      dest[c0 + _LAT_2C_] += b_y_recv_vec[c0 + _LAT_1C_];
      dest[c0 + _LAT_3C_] -= b_y_recv_vec[c0];
    }
  }
  // just add
  add_dest(origin_dest, dest, lat_tzyx);
  for (int i = 0; i < _LAT_SC_; i++) {
    dest[i] = zero;
  }
  {
    y = lat_y - 1; // f_y
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_y_recv_vec =
        ((static_cast<LatticeComplex *>(device_f_y_recv_vec)) +
         (((t * lat_z + z)) * lat_x + x));
  }
  { // y+1
    // move_forward(move, y, lat_y);
    // recv in y+1 way
    give_recv(f_y_recv_vec, origin_f_y_recv_vec, lat_tzyx / lat_y);
    tmp_U = (origin_U + (_Y_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    {
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += f_y_recv_vec[c1] * U[c0 * _LAT_C_ + c1];
          tmp1 += f_y_recv_vec[c1 + _LAT_1C_] * U[c0 * _LAT_C_ + c1];
        }
        dest[c0] += tmp0;
        dest[c0 + _LAT_1C_] += tmp1;
        dest[c0 + _LAT_2C_] -= tmp1;
        dest[c0 + _LAT_3C_] += tmp0;
      }
    }
  } // just add
  add_dest(origin_dest, dest, lat_tzyx);
#endif
}
__global__ void wilson_dslash_z_send(void *device_U, void *device_src,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_z_send_vec,
                                     void *device_f_z_send_vec) {
#ifdef __Z__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  int lat_y = lat_xyzt[_Y_];
  // int lat_z = zzztsc[_z_];
  int lat_z = 1; // so let z=0 first, then z = lat_z -1
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_z_send_vec[_LAT_HALF_SC_];
  LatticeComplex f_z_send_vec[_LAT_HALF_SC_];
  LatticeComplex *origin_U;
  LatticeComplex *origin_src;
  LatticeComplex *origin_b_z_send_vec;
  LatticeComplex *origin_f_z_send_vec;
  {
    lat_z = lat_xyzt[_Z_]; // give lat_size back
    z = 0;                 // b_z
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_z_send_vec =
        ((static_cast<LatticeComplex *>(device_b_z_send_vec)) +
         (((t)*lat_y + y) * lat_x + x));
  }
  { // z-1
    // move_backward(move, z, lat_z);
    // send in z+1 way
    give_src(src, origin_src, lat_tzyx);
    { // sigma src
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        b_z_send_vec[c1] = src[c1] - src[c1 + _LAT_2C_] * I;
        b_z_send_vec[c1 + _LAT_1C_] =
            src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_] * I;
      }
      give_send(origin_b_z_send_vec, b_z_send_vec, lat_tzyx / lat_z);
    }
  }
  {
    z = lat_z - 1; // f_z
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_z_send_vec =
        ((static_cast<LatticeComplex *>(device_f_z_send_vec)) +
         (((t)*lat_y + y) * lat_x + x));
  }
  { // z+1
    // move_forward(move, z, lat_z);
    // send in z-1 way
    tmp_U =
        (origin_U + (_Z_ * _EVENODD_ + (1 - parity)) * lat_tzyx); // even-odd
    give_u(U, tmp_U, lat_tzyx);
    give_src(src, origin_src, lat_tzyx);
    { // just tmp
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 +=
              (src[c1] + src[c1 + _LAT_2C_] * I) * U[c1 * _LAT_C_ + c0].conj();
          tmp1 += (src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_] * I) *
                  U[c1 * _LAT_C_ + c0].conj();
        }
        f_z_send_vec[c0] = tmp0;
        f_z_send_vec[c0 + _LAT_1C_] = tmp1;
      }
      give_send(origin_f_z_send_vec, f_z_send_vec, lat_tzyx / lat_z);
    }
  }
#endif
}
__global__ void wilson_dslash_z_recv(void *device_U, void *device_dest,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_z_recv_vec,
                                     void *device_f_z_recv_vec) {
#ifdef __Z__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  int lat_y = lat_xyzt[_Y_];
  // int lat_z = zzztsc[_z_];
  int lat_z = 1; // so let z=0 first, then z = lat_z -1
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_z_recv_vec[_LAT_HALF_SC_];
  LatticeComplex f_z_recv_vec[_LAT_HALF_SC_]; // needed
  LatticeComplex *origin_U;
  LatticeComplex *origin_dest;
  LatticeComplex *origin_b_z_recv_vec;
  LatticeComplex *origin_f_z_recv_vec;
  {
    lat_z = lat_xyzt[_Z_]; // give lat_size back
    z = 0;                 // b_z
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_z_recv_vec =
        ((static_cast<LatticeComplex *>(device_b_z_recv_vec)) +
         (((t)*lat_y + y) * lat_x + x));
  }
  { // z-1
    // move_backward(move, z, lat_z);
    // recv in z-1 way
    give_recv(b_z_recv_vec, origin_b_z_recv_vec, lat_tzyx / lat_z);
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      dest[c0] += b_z_recv_vec[c0];
      dest[c0 + _LAT_1C_] += b_z_recv_vec[c0 + _LAT_1C_];
      dest[c0 + _LAT_2C_] -= b_z_recv_vec[c0] * I;
      dest[c0 + _LAT_3C_] += b_z_recv_vec[c0 + _LAT_1C_] * I;
    }
  }
  // just add
  add_dest(origin_dest, dest, lat_tzyx);
  for (int i = 0; i < _LAT_SC_; i++) {
    dest[i] = zero;
  }
  {
    z = lat_z - 1; // f_z
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_z_recv_vec =
        ((static_cast<LatticeComplex *>(device_f_z_recv_vec)) +
         (((t)*lat_y + y) * lat_x + x));
  }
  { // z+1
    // move_forward(move, z, lat_z);
    // recv in z+1 way
    give_recv(f_z_recv_vec, origin_f_z_recv_vec, lat_tzyx / lat_z);
    tmp_U = (origin_U + (_Z_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    {
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += f_z_recv_vec[c1] * U[c0 * _LAT_C_ + c1];
          tmp1 += f_z_recv_vec[c1 + _LAT_1C_] * U[c0 * _LAT_C_ + c1];
        }
        dest[c0] += tmp0;
        dest[c0 + _LAT_1C_] += tmp1;
        dest[c0 + _LAT_2C_] += tmp0 * I;
        dest[c0 + _LAT_3C_] -= tmp1 * I;
      }
    }
  } // just add
  add_dest(origin_dest, dest, lat_tzyx);
#endif
}
__global__ void wilson_dslash_t_send(void *device_U, void *device_src,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_t_send_vec,
                                     void *device_f_t_send_vec) {
#ifdef __T__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  int lat_y = lat_xyzt[_Y_];
  int lat_z = lat_xyzt[_Z_];
  // int lat_t = ttttsc[_t_];
  int lat_t = 1; // so let t=0 first, then t = lat_t -1
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex src[_LAT_SC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_t_send_vec[_LAT_HALF_SC_];
  LatticeComplex f_t_send_vec[_LAT_HALF_SC_];
  LatticeComplex *origin_U;
  LatticeComplex *origin_src;
  LatticeComplex *origin_b_t_send_vec;
  LatticeComplex *origin_f_t_send_vec;
  {
    lat_t = lat_xyzt[_T_]; // give lat_size back
    t = 0;                 // b_t
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_t_send_vec =
        ((static_cast<LatticeComplex *>(device_b_t_send_vec)) +
         (((z)*lat_y + y) * lat_x + x));
  }
  { // t-1
    // move_backward(move, t, lat_t);
    // send in t+1 way
    give_src(src, origin_src, lat_tzyx);
    { // sigma src
      for (int c1 = 0; c1 < _LAT_C_; c1++) {
        b_t_send_vec[c1] = src[c1] - src[c1 + _LAT_2C_];
        b_t_send_vec[c1 + _LAT_1C_] = src[c1 + _LAT_1C_] - src[c1 + _LAT_3C_];
      }
      give_send(origin_b_t_send_vec, b_t_send_vec, lat_tzyx / lat_t);
    }
  }
  {
    t = lat_t - 1; // f_t
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_src = ((static_cast<LatticeComplex *>(device_src)) +
                  (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_t_send_vec =
        ((static_cast<LatticeComplex *>(device_f_t_send_vec)) +
         (((z)*lat_y + y) * lat_x + x));
  }
  { // t+1
    // move_forward(move, t, lat_t);
    // send in t-1 way
    tmp_U =
        (origin_U + (_T_ * _EVENODD_ + (1 - parity)) * lat_tzyx); // even-odd
    give_u(U, tmp_U, lat_tzyx);
    give_src(src, origin_src, lat_tzyx);
    { // just tmp
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += (src[c1] + src[c1 + _LAT_2C_]) * U[c1 * _LAT_C_ + c0].conj();
          tmp1 += (src[c1 + _LAT_1C_] + src[c1 + _LAT_3C_]) *
                  U[c1 * _LAT_C_ + c0].conj();
        }
        f_t_send_vec[c0] = tmp0;
        f_t_send_vec[c0 + _LAT_1C_] = tmp1;
      }
      give_send(origin_f_t_send_vec, f_t_send_vec, lat_tzyx / lat_t);
    }
  }
#endif
}
__global__ void wilson_dslash_t_recv(void *device_U, void *device_dest,
                                     void *device_lat_xyzt, int device_parity,
                                     void *device_b_t_recv_vec,
                                     void *device_f_t_recv_vec) {
#ifdef __T__
  int parity = blockIdx.x * blockDim.x + threadIdx.x;
  int *lat_xyzt = static_cast<int *>(device_lat_xyzt);
  int lat_x = lat_xyzt[_X_];
  int lat_y = lat_xyzt[_Y_];
  int lat_z = lat_xyzt[_Z_];
  // int lat_t = ttttsc[_t_];
  int lat_t = 1; // so let t=0 first, then t = lat_t -1
  int lat_tzyx = lat_xyzt[_XYZT_];
  int move;
  move = lat_x * lat_y * lat_z;
  int t = parity / move;
  parity -= t * move;
  move = lat_x * lat_y;
  int z = parity / move;
  parity -= z * move;
  int y = parity / lat_x;
  int x = parity - y * lat_x;
  parity = device_parity;
  LatticeComplex I(0.0, 1.0);
  LatticeComplex zero(0.0, 0.0);
  LatticeComplex *tmp_U;
  LatticeComplex tmp0(0.0, 0.0);
  LatticeComplex tmp1(0.0, 0.0);
  LatticeComplex U[_LAT_CC_];
  LatticeComplex dest[_LAT_SC_];
  LatticeComplex b_t_recv_vec[_LAT_HALF_SC_];
  LatticeComplex f_t_recv_vec[_LAT_HALF_SC_]; // needed
  LatticeComplex *origin_U;
  LatticeComplex *origin_dest;
  LatticeComplex *origin_b_t_recv_vec;
  LatticeComplex *origin_f_t_recv_vec;
  {
    lat_t = lat_xyzt[_T_]; // give lat_size back
    t = 0;                 // b_t
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_b_t_recv_vec =
        ((static_cast<LatticeComplex *>(device_b_t_recv_vec)) +
         (((z)*lat_y + y) * lat_x + x));
  }
  { // t-1
    // move_backward(move, t, lat_t);
    // recv in t-1 way
    give_recv(b_t_recv_vec, origin_b_t_recv_vec, lat_tzyx / lat_t);
    for (int c0 = 0; c0 < _LAT_C_; c0++) {
      dest[c0] += b_t_recv_vec[c0];
      dest[c0 + _LAT_1C_] += b_t_recv_vec[c0 + _LAT_1C_];
      dest[c0 + _LAT_2C_] += b_t_recv_vec[c0];
      dest[c0 + _LAT_3C_] += b_t_recv_vec[c0 + _LAT_1C_];
    }
  }
  // just add
  add_dest(origin_dest, dest, lat_tzyx);
  for (int i = 0; i < _LAT_SC_; i++) {
    dest[i] = zero;
  }
  {
    t = lat_t - 1; // f_t
    origin_U = ((static_cast<LatticeComplex *>(device_U)) +
                (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_dest = ((static_cast<LatticeComplex *>(device_dest)) +
                   (((t * lat_z + z) * lat_y + y) * lat_x + x));
    origin_f_t_recv_vec =
        ((static_cast<LatticeComplex *>(device_f_t_recv_vec)) +
         (((z)*lat_y + y) * lat_x + x));
  }
  { // t+1
    // move_forward(move, t, lat_t);
    // recv in t+1 way
    give_recv(f_t_recv_vec, origin_f_t_recv_vec, lat_tzyx / lat_t);
    tmp_U = (origin_U + (_T_ * _EVENODD_ + parity) * lat_tzyx);
    give_u(U, tmp_U, lat_tzyx);
    {
      for (int c0 = 0; c0 < _LAT_C_; c0++) {
        tmp0 = zero;
        tmp1 = zero;
        for (int c1 = 0; c1 < _LAT_C_; c1++) {
          tmp0 += f_t_recv_vec[c1] * U[c0 * _LAT_C_ + c1];
          tmp1 += f_t_recv_vec[c1 + _LAT_1C_] * U[c0 * _LAT_C_ + c1];
        }
        dest[c0] += tmp0;
        dest[c0 + _LAT_1C_] += tmp1;
        dest[c0 + _LAT_2C_] -= tmp0;
        dest[c0 + _LAT_3C_] -= tmp1;
      }
    }
  } // just add
  add_dest(origin_dest, dest, lat_tzyx);
#endif
}
#endif