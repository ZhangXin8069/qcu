#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "comm/qcu_communicator.h"
#include "mempool/qcu_mempool.h"
#include "qcd/qcu_wilson_dslash.cuh"
#include "qcu.h"
#include "qcu_macro.cuh"
#include "qcu_storage/qcu_storage.cuh"
#include "solver/qcu_cg.h"
// #define PRINT_EXEC_TIME
// #define PRINT_ALLOCATED_MEM_SIZE

BEGIN_NAMESPACE(qcu)

class Qcu {
 protected:
  bool gaugeLoaded_;

  double kappa_;
  double mass_;

  int Lx_;
  int Ly_;
  int Lz_;
  int Lt_;

  int procNx_;
  int procNy_;
  int procNz_;
  int procNt_;
  int boundaryLength_[Nd];

  void *inputGauge_;

  void *coalescedGauge_;       // coalesced gauge
  void *coalescedFermionIn_;   // coalesced fermion
  void *coalescedFermionOut_;  // coalesced fermion

  void *fermionIn_;   // input fermion
  void *fermionOut_;  // output fermion

  void *cloverMatrix_;
  void *cloverInvMatrix_;

  hipStream_t stream1_;
  hipStream_t stream2_;
  hipStream_t commStreams_[Nd * DIRECTIONS];
  hipEvent_t startEvent_;
  hipEvent_t stopEvent_;

  QcuMemPool *memPool_;
  MsgHandler *msgHandler_;
  QcuComm *qcuComm_;

 public:
  Qcu(int Lx, int Ly, int Lz, int Lt, int Nx, int Ny, int Nz, int Nt, double mass = 0.0)
      : Lx_(Lx),
        Ly_(Ly),
        Lz_(Lz),
        Lt_(Lt),
        procNx_(Nx),
        procNy_(Ny),
        procNz_(Nz),
        procNt_(Nt),
        mass_(mass),
        kappa_(1.0 / (2.0 * (4.0 + mass))),
        gaugeLoaded_(false),
        inputGauge_(nullptr),
        coalescedGauge_(nullptr),
        coalescedFermionIn_(nullptr),
        coalescedFermionOut_(nullptr),
        fermionIn_(nullptr),
        fermionOut_(nullptr),
        cloverMatrix_(nullptr),
        cloverInvMatrix_(nullptr),
        memPool_(nullptr),
        msgHandler_(nullptr),
        qcuComm_(nullptr) {
    CHECK_CUDA(hipStreamCreate(&stream1_));
    CHECK_CUDA(hipStreamCreate(&stream2_));
    for (int i = 0; i < Nd * DIRECTIONS; i++) {
      CHECK_CUDA(hipStreamCreate(&commStreams_[i]));
    }

    CHECK_CUDA(hipEventCreate(&startEvent_));
    CHECK_CUDA(hipEventCreate(&stopEvent_));
    int vol = Lx_ * Ly_ * Lz_ * Lt_;

    // TODO：此部分分配内存过剩，看一下怎么减少
    CHECK_CUDA(hipMalloc(&coalescedFermionIn_, sizeof(double) * 2 * vol * Ns * Nc));
    CHECK_CUDA(hipMalloc(&coalescedFermionOut_, sizeof(double) * 2 * vol * Ns * Nc));
    msgHandler_ = new MsgHandler();
    qcuComm_ = new QcuComm(procNx_, procNy_, procNz_, procNt_);
    memPoolInit();
#ifdef MPI_START_SENDRECV
    initSendRecv();
#endif

#ifdef DEBUG
    printf("Qcu mass = %lf, kappa_ = %lf\n", mass_, kappa_);
#endif
  }
  virtual ~Qcu() {
    CHECK_CUDA(hipStreamDestroy(stream1_));
    CHECK_CUDA(hipStreamDestroy(stream2_));
    for (int i = 0; i < Nd * DIRECTIONS; i++) {
      CHECK_CUDA(hipStreamDestroy(commStreams_[i]));
    }

    CHECK_CUDA(hipEventDestroy(startEvent_));
    CHECK_CUDA(hipEventDestroy(stopEvent_));

    if (coalescedGauge_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedGauge_));
      coalescedGauge_ = nullptr;
    }
    if (coalescedFermionIn_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedFermionIn_));
      coalescedFermionIn_ = nullptr;
    }
    if (coalescedFermionOut_ != nullptr) {
      CHECK_CUDA(hipFree(coalescedFermionOut_));
      coalescedFermionOut_ = nullptr;
    }
    if (memPool_ != nullptr) {
      delete memPool_;
      memPool_ = nullptr;
    }
    if (msgHandler_ != nullptr) {
      delete msgHandler_;
      msgHandler_ = nullptr;
    }
    if (qcuComm_ != nullptr) {
      delete qcuComm_;
      qcuComm_ = nullptr;
    }
  }
  void memPoolInit() {
    memPool_ = new QcuMemPool();
    int singleVecLength = Ns * Nc;

    boundaryLength_[0] = procNx_ == 1 ? 0 : Ly_ * Lz_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[1] = procNy_ == 1 ? 0 : Lx_ * Lz_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[2] = procNz_ == 1 ? 0 : Lx_ * Ly_ * Lt_ / 2 * singleVecLength;
    boundaryLength_[3] = procNt_ == 1 ? 0 : Lx_ * Ly_ * Lz_ / 2 * singleVecLength;

    memPool_->allocateAllVector(boundaryLength_[0], boundaryLength_[1], boundaryLength_[2], boundaryLength_[3],
                                sizeof(double) * 2);
#ifdef PRINT_ALLOCATED_MEM_SIZE
    printf("========================\n");
    printf("Allocated memory size : \n x dim = %d\n y dim = %d\n z dim = %d\n t dim = %d\n ", boundaryLength_[0],
           boundaryLength_[1], boundaryLength_[2], boundaryLength_[3]);
    printf("========================\n");
#endif
  }

  void loadGauge(void *gauge);
  void shiftFermionStorage(void *dst, void *src, int shiftDir);

  virtual void wilsonDslashMultiProc(void *fermionOut, void *fermionIn, int parity);
  virtual void qcuInvert(void *fermionOutX, void *fermionInB, double diffTarget, int maxIterations);
  virtual void initSendRecv();
};

void Qcu::wilsonDslashMultiProc(void *fermionOut, void *fermionIn, int parity) {
  // shiftStorage
  int daggerFlag = 0;
  fermionIn_ = fermionIn;
  fermionOut_ = fermionOut;
  shiftFermionStorage(coalescedFermionIn_, fermionIn_, TO_COALESCE);

  DslashParam dslashParam(coalescedFermionIn_, coalescedFermionOut_, coalescedGauge_, Lx_, Ly_, Lz_, Lt_, parity,
                          procNx_, procNy_, procNz_, procNt_, kappa_, daggerFlag, memPool_, msgHandler_, qcuComm_,
                          stream1_, stream2_, commStreams_);
  CHECK_CUDA(hipEventRecord(startEvent_, stream1_));
  WilsonDslash dslash(&dslashParam, 256);

  // dslash.preApply();
  // dslash.apply();
  // dslash.postApply();
  // dslash.preApply2();
  // dslash.apply();
  // dslash.postApply2();

  DslashMV dslashMv(&dslash);
  dslashMv(coalescedFermionOut_, coalescedFermionIn_);
  CHECK_CUDA(hipEventRecord(stopEvent_, stream1_));
  CHECK_CUDA(hipEventSynchronize(stopEvent_));
#ifdef PRINT_EXEC_TIME
  float elapsedTime;
  CHECK_CUDA(hipEventElapsedTime(&elapsedTime, startEvent_, stopEvent_));
  printf("Recorded time : %f s\n", elapsedTime / 1000);
#endif
  // shiftStorage
  shiftFermionStorage(fermionOut_, coalescedFermionOut_, TO_NON_COALESCE);
}

void Qcu::loadGauge(void *gauge) {
  if (!gaugeLoaded_ && coalescedGauge_ == nullptr) {
    CHECK_CUDA(hipMalloc(&coalescedGauge_, sizeof(double) * Nd * Lx_ * Ly_ * Lz_ * Lt_ * (Nc - 1) * Nc * 2));
  }

  shiftGaugeStorageTwoDouble(coalescedGauge_, gauge, TO_COALESCE, Lx_, Ly_, Lz_, Lt_);
  gaugeLoaded_ = true;
}

// TODO : 消除不必要代码
void Qcu::shiftFermionStorage(void *dst, void *src, int shiftDir) {
  if (shiftDir == TO_COALESCE) {
    shiftVectorStorageTwoDouble(dst, src, TO_COALESCE, Lx_, Ly_, Lz_, Lt_);
  } else if (shiftDir == TO_NON_COALESCE) {
    shiftVectorStorageTwoDouble(dst, src, TO_NON_COALESCE, Lx_, Ly_, Lz_, Lt_);
  }
}

void Qcu::qcuInvert(void *fermionOutX, void *fermionInB, double diffTarget, int maxIterations) {
  int vol = Lx_ * Ly_ * Lz_ * Lt_;
  int halfVol = vol / 2;
  void *originFermionInEven = fermionInB;
  void *originFermionInOdd = static_cast<void *>(static_cast<Complex *>(fermionInB) + halfVol * Ns * Nc);
  void *originFermionOutEven = fermionOutX;
  void *originFermionOutOdd = static_cast<void *>(static_cast<Complex *>(fermionOutX) + halfVol * Ns * Nc);

  void *newFermionInEven = coalescedFermionIn_;
  void *newFermionInOdd = static_cast<void *>(static_cast<Complex *>(coalescedFermionIn_) + halfVol * Ns * Nc);
  void *newFermionOutEven = coalescedFermionOut_;
  void *newFermionOutOdd = static_cast<void *>(static_cast<Complex *>(coalescedFermionOut_) + halfVol * Ns * Nc);

  // shift storage to coalesce
  shiftFermionStorage(newFermionInEven, originFermionInEven, TO_COALESCE);
  shiftFermionStorage(newFermionInOdd, originFermionInOdd, TO_COALESCE);
#ifdef DEBUG
  QcuNorm2 norm2(msgHandler_, 256);
  void *temp1;
  void *temp2;
  double norm;
  CHECK_CUDA(hipMalloc(&temp1, sizeof(Complex) * vol * Ns * Nc));
  CHECK_CUDA(hipMalloc(&temp2, sizeof(Complex) * vol * Ns * Nc));
  norm2(temp1, temp2, fermionInB, vol * Ns * Nc);
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(&norm, temp1, sizeof(double), hipMemcpyDeviceToHost));
  printf("__FILE__ = %s, __LINE__ = %d, norm = %lf, addr(b) = %p\n", __FILE__, __LINE__, norm, fermionInB);
  norm2(temp1, temp2, newFermionInEven, vol * Ns * Nc);
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(&norm, temp1, sizeof(double), hipMemcpyDeviceToHost));
  printf("__FILE__ = %s, __LINE__ = %d, coalesced norm = %lf, addr(newB) = %p\n", __FILE__, __LINE__, norm,
         newFermionInEven);
#endif

  // CG Inverter
  // generate CGParam
  assert(coalescedGauge_ != nullptr);
  CGParam cgParam(coalescedFermionIn_, coalescedFermionOut_, coalescedGauge_, nullptr, nullptr, kappa_, Lx_, Ly_, Lz_,
                  Lt_, procNx_, procNy_, procNz_, procNt_, memPool_, msgHandler_, qcuComm_, stream1_, stream2_,
                  commStreams_);

  QcuCG qcuWilsonSolver_CG(DSLASH_WILSON, &cgParam, diffTarget, maxIterations, 256);
  qcuWilsonSolver_CG.qcuInvert();
  // shift back
  shiftFermionStorage(originFermionOutEven, newFermionOutEven, TO_NON_COALESCE);
  shiftFermionStorage(originFermionOutOdd, newFermionOutOdd, TO_NON_COALESCE);
}

// 假设msgHandler已经申请完毕
void Qcu::initSendRecv() {
  assert(msgHandler_ != nullptr);
  int fwdRank;
  int bwdRank;
  void *sendBufFWD;
  void *sendBufBWD;
  void *recvBufFWD;
  void *recvBufBWD;
  int commVecLength[Nd] = {0, 0, 0, 0};
  if (procNx_ > 1) {
    commVecLength[X_DIM] = Ly_ * Lz_ * Lt_ / 2 * Ns * Nc;
  }
  if (procNy_ > 1) {
    commVecLength[Y_DIM] = Lx_ * Lz_ * Lt_ / 2 * Ns * Nc;
  }
  if (procNz_ > 1) {
    commVecLength[Z_DIM] = Lx_ * Ly_ * Lt_ / 2 * Ns * Nc;
  }
  if (procNt_ > 1) {
    commVecLength[T_DIM] = Lx_ * Ly_ * Lz_ / 2 * Ns * Nc;
  }

  for (int dim = X_DIM; dim < Nd; dim++) {
    if (commVecLength[dim] == 0) { // commVecLength[dim] == 0时说明这个方向不通信，没有必要sendInit
      continue;
    }
    fwdRank = qcuComm_->getNeighborRank(dim, FWD);
    bwdRank = qcuComm_->getNeighborRank(dim, BWD);

    sendBufFWD = memPool_->h_send_buffer[dim][FWD];
    sendBufBWD = memPool_->h_send_buffer[dim][BWD];
    msgHandler_->msgSendInit(dim, fwdRank, bwdRank, commVecLength[dim], sendBufFWD, sendBufBWD);

    recvBufFWD = memPool_->h_recv_buffer[dim][FWD];
    recvBufBWD = memPool_->h_recv_buffer[dim][BWD];
    msgHandler_->msgRecvInit(dim, fwdRank, bwdRank, commVecLength[dim], recvBufFWD, recvBufBWD);
  }
}
END_NAMESPACE(qcu)

static qcu::Qcu *qcu_ptr = nullptr;

void initGridSize(QcuGrid_t *grid, QcuParam *p_param, void *gauge, void *fermion_in, void *fermion_out) {
  if (qcu_ptr == nullptr) {
    qcu_ptr = new qcu::Qcu(p_param->lattice_size[0], p_param->lattice_size[1], p_param->lattice_size[2],
                           p_param->lattice_size[3], grid->grid_size[0], grid->grid_size[1], grid->grid_size[2],
                           grid->grid_size[3]);
  }
}

void destroyQcu() {
  if (qcu_ptr != nullptr) {
    delete qcu_ptr;
    qcu_ptr = nullptr;
  }
}

void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int parity) {
  // qcu_ptr->loadGauge(gauge);
  qcu_ptr->wilsonDslashMultiProc(fermion_out, fermion_in, parity);
}

void fullDslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param, int dagger_flag) {}
void cg_inverter(void *x_vector, void *b_vector, void *gauge, QcuParam *param, double p_max_prec, double p_kappa) {
  // p_kappa is useless
  // qcu_ptr->qcuInvert(x_vector, b_vector, p_max_prec);
  qcu_ptr->qcuInvert(x_vector, b_vector, p_max_prec, 1000);
}

void loadQcuGauge(void *gauge, QcuParam *param) { qcu_ptr->loadGauge(gauge); }