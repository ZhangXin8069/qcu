#include "hip/hip_runtime.h"
#include "qcu_shift_storage.cuh"
#include <cstdio>
#include "qcu_macro.cuh"

// DONE: WARP version, no sync  
static __device__ void storeVectorBySharedMemory(void* origin, void* result) {
  __shared__ double shared_buffer[BLOCK_SIZE * Ns * Nc * 2];
  int thread = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_index = (thread - thread / BLOCK_SIZE * BLOCK_SIZE) / WARP_SIZE;//thread % BLOCK_SIZE / WARP_SIZE;

  // result is register variable
  double* shared_dst = shared_buffer + threadIdx.x * Ns * Nc * 2;
  double* warp_src = static_cast<double*>(origin) + (thread / WARP_SIZE * WARP_SIZE) * Ns * Nc * 2;


  // load data to register
  double* register_addr = static_cast<double*>(result);
  for (int i = 0; i < Ns * Nc * 2; i++) {
    shared_dst[i] = register_addr[i];
  }

  // store result of shared memory to global memory
  for (int i = threadIdx.x - threadIdx.x / WARP_SIZE * WARP_SIZE; i < WARP_SIZE * Ns * Nc * 2; i += WARP_SIZE) {
    warp_src[i] = shared_buffer[warp_index * WARP_SIZE * Ns * Nc * 2 + i];
  }
}

// DONE: WARP version, no sync  
static __device__ void loadVectorBySharedMemory(void* origin, void* result) {
  __shared__ double shared_buffer[BLOCK_SIZE * Ns * Nc * 2];
  int thread = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_index = (thread - thread / BLOCK_SIZE * BLOCK_SIZE) / WARP_SIZE;//thread % BLOCK_SIZE / WARP_SIZE;

  // result is register variable
  double* shared_dst = shared_buffer + threadIdx.x * Ns * Nc * 2;
  double* warp_src = static_cast<double*>(origin) + (thread / WARP_SIZE * WARP_SIZE) * Ns * Nc * 2;

  // store result of shared memory to global memory
  for (int i = threadIdx.x - threadIdx.x / WARP_SIZE * WARP_SIZE; i < WARP_SIZE * Ns * Nc * 2; i += WARP_SIZE) {
    shared_buffer[warp_index * WARP_SIZE * Ns * Nc * 2 + i] = warp_src[i];
  }

  // load data to register
  double* register_addr = static_cast<double*>(result);
  for (int i = 0; i < Ns * Nc * 2; i++) {
    register_addr[i] = shared_dst[i];
  }
}


static __device__ void loadGaugeBySharedMemory(void* origin, void* result) {
  __shared__ double shared_buffer[BLOCK_SIZE * Nc * Nc * 2];
  int thread = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_index = (thread - thread / BLOCK_SIZE * BLOCK_SIZE) / WARP_SIZE;//thread % BLOCK_SIZE / WARP_SIZE;

  // result is register variable
  double* shared_dst = shared_buffer + threadIdx.x * Nc * Nc * 2;
  double* warp_src = static_cast<double*>(origin) + (thread / WARP_SIZE * WARP_SIZE) * Nc * Nc * 2;

  // store result of shared memory to global memory
  for (int i = threadIdx.x - threadIdx.x / WARP_SIZE * WARP_SIZE; i < WARP_SIZE * Nc * Nc * 2; i += WARP_SIZE) {
    shared_buffer[warp_index * WARP_SIZE * Nc * Nc * 2 + i] = warp_src[i];
  }

  // load data to register
  double* register_addr = static_cast<double*>(result);
  for (int i = 0; i < (Nc-1) * Nc * 2; i++) {
    register_addr[i] = shared_dst[i];
  }
}


// DONE: Lx is full Lx, not Lx / 2
static __global__ void shift_vector_to_coalesed (void* dst_vec, void* src_vec, int Lx, int Ly, int Lz, int Lt) {
  // change storage to [parity, Ns, Nc, 2, t, z, y, x]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  // double* src_vec_pointer = static_cast<double*>(src_vec) + thread_id * Ns * Nc * 2;
  double* dst_vec_pointer = static_cast<double*>(dst_vec) + thread_id;

  // mofify 
  double data_local[Ns * Nc * 2];
  loadVectorBySharedMemory(src_vec, data_local);
  for (int i = 0; i < Ns * Nc * 2; i++) {
    *dst_vec_pointer = data_local[i];
    dst_vec_pointer += sub_vol;
  }
}


// TODO: TO optimize      Lx is full Lx, not Lx / 2
static __global__ void shift_vector_to_noncoalesed (void* dst_vec, void* src_vec, int Lx, int Ly, int Lz, int Lt) {
  // change storage to [parity, Ns, Nc, 2, t, z, y, x]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  // double* dst_vec_pointer = static_cast<double*>(dst_vec) + thread_id * Ns * Nc * 2;
  double* src_vec_pointer = static_cast<double*>(src_vec) + thread_id;

  // for (int i = 0; i < Ns * Nc * 2; i++) {
  //   dst_vec_pointer[i] = *src_vec_pointer;
  //   src_vec_pointer += sub_vol;
  // }
  double local_vector[Ns * Nc * 2];
  for (int i = 0; i < Ns * Nc * 2; i++) {
    // *dst_vec_pointer = data_local[i];
    // dst_vec_pointer += sub_vol;
    local_vector[i] = *src_vec_pointer;
    src_vec_pointer += sub_vol;
  }
  storeVectorBySharedMemory(dst_vec, local_vector);

}





// Lx is full Lx, not Lx / 2
static __global__ void shift_gauge_to_coalesed (void* dst_gauge, void* src_gauge, int Lx, int Ly, int Lz, int Lt) {
  // each thread shift both even and odd part
  // change storage to [Nd, parity, Nc-1, Nc, 2, t, z, y, x/2]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  int t = thread_id / (Lz * Ly * sub_Lx);
  int z = thread_id % (Lz * Ly * sub_Lx) / (Ly * sub_Lx);
  int y = thread_id % (Ly * sub_Lx) / sub_Lx;
  int sub_x = thread_id % sub_Lx;

  double local_gauge[(Nc - 1) * Nc * 2];

  double* dst_gauge_ptr;
  double* src_gauge_ptr;
  for (int i = 0; i < Nd; i++) {
    for (int parity = 0; parity < 2; parity++) {
      // dst_gauge_ptr = static_cast<double*>(dst_gauge) + (2 * i + parity) * sub_vol * (Nc - 1) * Nc * 2 + (((t * Lz + z) * Ly + y) * sub_Lx + sub_x);
      // src_gauge_ptr = static_cast<double*>(src_gauge) + (2 * i + parity) * sub_vol * Nc * Nc * 2 + (((t * Lz + z) * Ly + y) * sub_Lx + sub_x) * 2 * Nc * Nc;

      // for (int j = 0 ; j < Nc * (Nc-1) * 2; j++) {
      //   // dst_gauge_ptr[i * Nc * (Nc - 1) + thread_id] = src_gauge_ptr[thread_id * Nc * Nc + i];
      //   *dst_gauge_ptr = *src_gauge_ptr;
      //   dst_gauge_ptr += sub_vol;
      //   src_gauge_ptr++;
      // }
      dst_gauge_ptr = static_cast<double*>(dst_gauge) + (2 * i + parity) * sub_vol * (Nc - 1) * Nc * 2 + (((t * Lz + z) * Ly + y) * sub_Lx + sub_x);
      src_gauge_ptr = static_cast<double*>(src_gauge) + (2 * i + parity) * sub_vol * Nc * Nc * 2; //  + (((t * Lz + z) * Ly + y) * sub_Lx + sub_x) * 2 * Nc * Nc;
      loadGaugeBySharedMemory(src_gauge_ptr, local_gauge);

      for (int j = 0 ; j < Nc * (Nc-1) * 2; j++) {
        *dst_gauge_ptr = local_gauge[j];
        dst_gauge_ptr += sub_vol;
      }
      
    }
  }
}




void shiftVectorStorage(void* dst_vec, void* src_vec, int shift_direction, int Lx, int Ly, int Lz, int Lt) {
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol / 2;

  int block_size = BLOCK_SIZE;
  int grid_size = (half_vol + block_size - 1) / block_size;

  if (shift_direction == TO_COALESCE) {
    shift_vector_to_coalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  } else {
    shift_vector_to_noncoalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  }
}


void shiftGaugeStorage(void* dst_vec, void* src_vec, int shift_direction, int Lx, int Ly, int Lz, int Lt) {
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol / 2;

  int block_size = BLOCK_SIZE;
  int grid_size = (half_vol + block_size - 1) / block_size;

  if (shift_direction == TO_COALESCE) {
    shift_gauge_to_coalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  }
}