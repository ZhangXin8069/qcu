#include "hip/hip_runtime.h"
#include "qcu_macro.cuh"
#include "qcu_shift_storage.cuh"



// Lx is full Lx, not Lx / 2
static __global__ shift_vector_to_coalesed (void* dst_vec, void* src_vec, int Lx, int Ly, int Lz, int Lt) {
  // change storage to [parity, Ns, Nc, 2, t, z, y, x]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  double* src_vec_pointer = static_cast<double*>(src_vec) + thread_id * Ns * Nc * 2;
  double* dst_vec_pointer = static_cast<double*>(dst_vec) + thread_id;

  for (int i = 0; i < Ns * Nc * 2; i++) {
    *dst_vec_pointer = src_vec_pointer[i];
    dst_vec_pointer += sub_vol;
  }
}

// Lx is full Lx, not Lx / 2
static __global__ shift_vector_to_noncoalesed (void* dst_vec, void* src_vec, int Lx, int Ly, int Lz, int Lt) {
  // change storage to [parity, Ns, Nc, 2, t, z, y, x]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  double* dst_vec_pointer = static_cast<double*>(src_vec) + thread_id * Ns * Nc * 2;
  double* src_vec_pointer = static_cast<double*>(dst_vec) + thread_id;

  for (int i = 0; i < Ns * Nc * 2; i++) {
    dst_vec_pointer[i] = *src_vec_pointer;
    src_vec_pointer += sub_vol;
  }
}


// Lx is full Lx, not Lx / 2
static __global__ shift_gauge_to_coalesed (void* dst_gauge, void* src_gauge, int Lx, int Ly, int Lz, int Lt) {
  // change storage to [parity, Ns, Nc, 2, t, z, y, x]
  int sub_Lx = Lx >> 1;
  int sub_vol = sub_Lx * Ly * Lz * Lt;
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  double* dst_gauge_ptr;
  double* src_gauge_ptr;
  for (int i = 0; i < Nd; i++) {
    dst_gauge_ptr = static_cast<double*>(dst_gauge) + 2 * sub_vol * Nc * Nc;
    src_gauge_ptr = static_cast<double*>(src_gauge) + 2 * sub_vol * Nc * Nc;

    for (int i = 0 ; i < Nc * (Nc-1) * 2; i++) {
      dst_gauge_ptr[i * Nc * (Nc - 1) + thread_id] = src_gauge_ptr[thread_id * Nc * Nc + i];
    }
  }
}

// Lx is full Lx, not Lx / 2
// static __global__ shift_gauge_to_noncoalesed (void* dst_gauge, void* src_gauge, int Lx, int Ly, int Lz, int Lt) {
//   // change storage to [parity, Ns, Nc, 2, t, z, y, x]
//   int sub_Lx = Lx >> 1;
//   int sub_vol = sub_Lx * Ly * Lz * Lt;
//   int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

//   double* dst_gauge_ptr;
//   double* src_gauge_ptr;
//   for (int i = 0; i < Nd; i++) {
//     dst_gauge_ptr = static_cast<double*>(dst_gauge) + 2 * sub_vol * Nc * Nc;
//     src_gauge_ptr = static_cast<double*>(src_gauge) + 2 * sub_vol * Nc * Nc;

//     for (int i = 0 ; i < Nc * (Nc-1) * 2; i++) {
//       dst_gauge_ptr[i * Nc * (Nc - 1) + thread_id] = src_gauge_ptr[thread_id * Nc * Nc + i];
//     }
//   }
// }



void shiftVectorStorage(void* dst_vec, void* src_vec, int shift_direction, int Lx, int Ly, int Lz, int Lt) {
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol / 2;

  int block_size = 256;
  int grid_size = (half_vol + block_size - 1) / block_size;

  if (shift_direction == TO_COALESCE) {
    shift_vector_to_coalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  } else {
    shift_vector_to_noncoalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  }
}

void shiftGaugeStorage(void* dst_vec, void* src_vec, int shift_direction, int Lx, int Ly, int Lz, int Lt) {
  int vol = Lx * Ly * Lz * Lt;
  int half_vol = vol / 2;

  int block_size = 256;
  int grid_size = (half_vol + block_size - 1) / block_size;

  if (shift_direction == TO_COALESCE) {
    shift_gauge_to_coalesed <<<grid_size, block_size>>>(dst_vec, src_vec, Lx, Ly, Lz, Lt);
    checkCudaErrors(hipDeviceSynchronize());
  }
}