#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE N * 20

__global__ void kernel(int *a, int *b, int *c) {
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadID < N) {
    c[threadID] = (a[threadID] + b[threadID]) / 2;
  }
}

int main() {
  // 获取设备属性
  hipDeviceProp_t prop;
  int deviceID;
  hipGetDevice(&deviceID);
  hipGetDeviceProperties(&prop, deviceID);

  // 检查设备是否支持重叠功能
  if (!prop.deviceOverlap) {
    printf("No device will handle overlaps. so no speed up from stream.\n");
    return 0;
  }

  // 启动计时器
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // 创建一个CUDA流
  hipStream_t stream;
  hipStreamCreate(&stream);

  int *host_a, *host_b, *host_c;
  int *dev_a, *dev_b, *dev_c;

  // 在GPU上分配内存
  hipMalloc((void **)&dev_a, N * sizeof(int));
  hipMalloc((void **)&dev_b, N * sizeof(int));
  hipMalloc((void **)&dev_c, N * sizeof(int));

  // 在CPU上分配页锁定内存
  hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
                hipHostMallocDefault);
  hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
                hipHostMallocDefault);
  hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
                hipHostMallocDefault);

  // 主机上的内存赋值
  for (int i = 0; i < FULL_DATA_SIZE; i++) {
    host_a[i] = i;
    host_b[i] = FULL_DATA_SIZE - i;
  }

  for (int i = 0; i < FULL_DATA_SIZE; i += N) {
    hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice,
                    stream);

    kernel<<<N / 1024, 1024, 0, stream>>>(dev_a, dev_b, dev_c);

    hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost,
                    stream);
  }

  // wait until gpu execution finish
  hipStreamSynchronize(stream);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  std::cout << "消耗时间： " << elapsedTime << std::endl;

  // 输出前10个结果
  for (int i = 0; i < 10; i++) {
    std::cout << host_c[i] << std::endl;
  }

  getchar();

  // free stream and mem
  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipStreamDestroy(stream);
  return 0;
}