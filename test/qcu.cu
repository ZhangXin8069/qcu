#include "./include/qcu.h"
#include "define.h"
int main() {
  MPI_Init(NULL, NULL);
  int param_lattice_size[_DIM_];
  int grid_lattice_size[_DIM_];
  for (int i = 0; i < _DIM_; i++) {
    param_lattice_size[i] = _LAT_EXAMPLE_;
    grid_lattice_size[i] = _GRID_EXAMPLE_;
  }
  grid_lattice_size[_T_] = 2;
  LatticeSet _set;
  _set.give(param_lattice_size, grid_lattice_size);
  _set.init();
  int parity = 1;
  void *gauge;
  void *fermion_in;
  void *fermion_out;
  checkCudaErrors(hipMalloc(
      &gauge, _LAT_DCC_ * _EVEN_ODD_ * _LAT_EXAMPLE_ * _LAT_EXAMPLE_ *
                  _LAT_EXAMPLE_ * _LAT_EXAMPLE_ * sizeof(LatticeComplex)));
  checkCudaErrors(hipStreamSynchronize(_set.stream));
  give_debug_u<<<_set.gridDim, _set.blockDim, 0, _set.stream>>>(
      gauge, _set.device_lat_xyzt, parity, _set.node_rank);
  checkCudaErrors(hipStreamSynchronize(_set.stream));
  checkCudaErrors(hipMalloc(
      &fermion_in, _LAT_SC_ * _EVEN_ODD_ * _LAT_EXAMPLE_ * _LAT_EXAMPLE_ *
                       _LAT_EXAMPLE_ * _LAT_EXAMPLE_ * sizeof(LatticeComplex)));
  checkCudaErrors(hipMalloc(&fermion_out, _LAT_SC_ * _EVEN_ODD_ *
                                               _LAT_EXAMPLE_ * _LAT_EXAMPLE_ *
                                               _LAT_EXAMPLE_ * _LAT_EXAMPLE_ *
                                               sizeof(LatticeComplex)));
  {
    // define for nccl_clover_dslash
    // LatticeSet _set;
    // _set.give(param_lattice_size, grid_lattice_size);
    // _set.init();
    if (_set.node_rank == 0)
      _set._print(); // test
    dptzyxcc2ccdptzyx(gauge, &_set);
    tzyxsc2sctzyx(fermion_in, &_set);
    tzyxsc2sctzyx(fermion_out, &_set);
    LatticeWilsonDslash _wilson_dslash;
    LatticeCloverDslash _clover_dslash;
    _wilson_dslash.give(&_set);
    _clover_dslash.give(&_set);
    _clover_dslash.init();
    {
      // wilson dslash
      _wilson_dslash.run_test(fermion_out, fermion_in, gauge, parity);
    }
    {
      // make clover
      _clover_dslash.make(gauge, parity);
    }
    {
      // inverse clover
      _clover_dslash.inverse();
    }
    {
      // give clover
      _clover_dslash.give(fermion_out);
    }
    ccdptzyx2dptzyxcc(gauge, &_set);
    sctzyx2tzyxsc(fermion_in, &_set);
    sctzyx2tzyxsc(fermion_out, &_set);
    _clover_dslash.end();
    // _set.end();
  }
  hipFree(gauge);
  hipFree(fermion_in);
  hipFree(fermion_out);
  _set.end();
  MPI_Finalize();
  return 0;
}